
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream> 
using namespace std; 

//const int threadAmount = 512;       //12345, 2m44s
//const int threadAmount = 256;   //12345, 2m28s
//const int threadAmount = 128;   //12345, 2m26s
//const int threadAmount = 64;   //12345, 2m27s
const int threadAmount = 32;   //12345, 3m17s
//const int threadAmount = 2;   //12345, 3m15s

__host__
double getUpperC(double a, int b){
    return pow((a-b), 1.0f/6) * 
           pow((a+b), 1.0f/6) *
           pow(a, 1.0f/6) *
           pow(a, 1.0f/6) *
           pow((a-b+(b/a)*b), 1.0f/6) *
           pow((a+b+(b/a)*b), 1.0f/6);
}

__host__
double getLowerC(double a, int b){
    return pow((a-b), 1.0f/6) * 
           pow((a+b), 1.0f/6) *
           pow(a, 1.0f/6) *
           pow(a, 1.0f/6) *
           pow((a-b+(b/a)*b), 1.0f/6) *
           pow((a+b+(b/a)*b), 1.0f/6)/pow(4, 1.0f/6);
}

__host__
double getUpperD(double a, int b, int c){
    return a * pow((1-(b/a)*(b/a)*(b/a)*(b/a)*(b/a)*(b/a)-(c/a)*(c/a)*(c/a)*(c/a)*(c/a)*(c/a)),1.0f/6);
}

__host__
double getLowerD(double a, int b, int c){
    return a * pow((1-(b/a)*(b/a)*(b/a)*(b/a)*(b/a)*(b/a)-(c/a)*(c/a)*(c/a)*(c/a)*(c/a)*(c/a)),1.0f/6)/pow(3,1.0f/6);
}

__host__
int host_getModOf6Power(uint64_t base, int mod){
    int exp = 6;
    int res = 1;
    while (exp > 0) {
       if (exp % 2 == 1)
          res= (res * base) % mod;
       exp = exp >> 1;
       base = (base * base) % mod;
       //base = ((base % mod) * (base % mod)) % mod;
    }
    return res;

}

__host__
int getLeftModSubIn3Numbers(int prime, int a, int b, int c){
    int result = host_getModOf6Power(a, prime) - host_getModOf6Power(b, prime) - host_getModOf6Power(c, prime);
    while (result < 0){
        result += prime;
    }
    return result;
}
__host__
bool isDecomposableIn3Numbers(int a, int b, int c){
    //bool result = true;
    //int primes[4] = {13,19,31,37};
    int tmp = getLeftModSubIn3Numbers(7,a,b,c);
    
    switch(tmp){  case 5:  case  6:return false;}
    if(getLeftModSubIn3Numbers(13,a,b,c) == 10)return false;

    tmp = getLeftModSubIn3Numbers(19,a,b,c);
    switch(tmp){ case  5 :  case  16 :  case 17: return false;}

    tmp = getLeftModSubIn3Numbers(31,a,b,c);
    //switch(tmp){  case 15 :  case 23 :  case  27 :  case  29:  case 30) return false;
    switch(tmp){
        case 15: case 23: case 27: case 29: case 30:
            return false;

    }

    tmp = getLeftModSubIn3Numbers(37,a,b,c);
    switch(tmp){  case 4:  case  5:  case  6:  case  7:  case  8:  case  14:  case  24:  case  34: 
        return false; 
    }
    return true;
}


/////////////////////////on Device
__device__
int getModOf6Power(uint64_t base, int mod){
    int exp = 6;
    int res = 1;
    while (exp > 0) {
       if (exp % 2 == 1)
          res= (res * base) % mod;
       exp = exp >> 1;
       base = (base * base) % mod;
       //base = ((base % mod) * (base % mod)) % mod;
    }
    return res;

}

__device__
double getUpperE(double a, int b, int c, int d){
    return a * pow((1-(b/a)*(b/a)*(b/a)*(b/a)*(b/a)*(b/a)-(c/a)*(c/a)*(c/a)*(c/a)*(c/a)*(c/a)-(d/a)*(d/a)*(d/a)*(d/a)*(d/a)*(d/a)), 1.0f/6);
}

__device__
double getLowerE(double a, int b, int c, int d){
    return a * pow((1-(b/a)*(b/a)*(b/a)*(b/a)*(b/a)*(b/a)-(c/a)*(c/a)*(c/a)*(c/a)*(c/a)*(c/a)-(d/a)*(d/a)*(d/a)*(d/a)*(d/a)*(d/a))/2, 1.0f/6);
}

__device__
double getUpperF(double a, int b, int c, int d, int e){
    return a * pow((1-(b/a)*(b/a)*(b/a)*(b/a)*(b/a)*(b/a)-(c/a)*(c/a)*(c/a)*(c/a)*(c/a)*(c/a)-(d/a)*(d/a)*(d/a)*(d/a)*(d/a)*(d/a)-(e/a)*(e/a)*(e/a)*(e/a)*(e/a)*(e/a)), 1.0f/6);
}

__device__
int getLeftModSubIn2Numbers(int prime, int a, int b, int c, int d){
    int result = getModOf6Power(a, prime) - getModOf6Power(b, prime) - getModOf6Power(c, prime) - getModOf6Power(d, prime);
    while (result < 0){
        result += prime;
    }
    return result;
}
__device__
int getLeftModSubIn1Numbers(int prime, int a, int b, int c, int d, int e){
    int result = getModOf6Power(a, prime) - getModOf6Power(b, prime) - getModOf6Power(c, prime) - getModOf6Power(d, prime) - getModOf6Power(e, prime);
    while (result < 0){
        result += prime;
    }
    return result;
}


__device__
bool isABCDEFModEqual(int a, int b, int c, int d, int e, int f){
    //there are 168 primes up to 1000
    //int primes1000[] = {2, 3, 5, 7, 11, 13, 17, 19, 23, 29, 31, 37, 41, 43, 47, 53, 59, 61, 67, 71, 73, 79, 83, 89, 97, 101, 103, 107, 109, 113, 127, 131, 137, 139, 149, 151, 157, 163, 167, 173, 179, 181, 191, 193, 197, 199, 211, 223, 227, 229, 233, 239, 241, 251, 257, 263, 269, 271, 277, 281, 283, 293, 307, 311, 313, 317, 331, 337, 347, 349, 353, 359, 367, 373, 379, 383, 389, 397, 401, 409, 419, 421, 431, 433, 439, 443, 449, 457, 461, 463, 467, 479, 487, 491, 499, 503, 509, 521, 523, 541, 547, 557, 563, 569, 571, 577, 587, 593, 599, 601, 607, 613, 617, 619, 631, 641, 643, 647, 653, 659, 661, 673, 677, 683, 691, 701, 709, 719, 727, 733, 739, 743, 751, 757, 761, 769, 773, 787, 797, 809, 811, 821, 823, 827, 829, 839, 853, 857, 859, 863, 877, 881, 883, 887, 907, 911, 919, 929, 937, 941, 947, 953, 967, 971, 977, 983, 991, 997};
    int primes1000[] = {2, 3, 5, 7};
    //for(int i = 0; i < 168; i++){
    for(int i = 0; i < 4; i++){
        int result = getModOf6Power(a, primes1000[i]) - getModOf6Power(b, primes1000[i]) - 
                    getModOf6Power(c, primes1000[i]) - getModOf6Power(d, primes1000[i]) - 
                     getModOf6Power(e, primes1000[i]) - getModOf6Power(f, primes1000[i]);
        // printf("getModOf6Power(a, primes1000[i]): %d\n", getModOf6Power(a, primes1000[i]));
        // printf("getModOf6Power(b, primes1000[i]): %d\n", getModOf6Power(b, primes1000[i]));
        // printf("getModOf6Power(c, primes1000[i]): %d\n", getModOf6Power(c, primes1000[i]));
        // printf("getModOf6Power(d, primes1000[i]): %d\n", getModOf6Power(d, primes1000[i]));
        // printf("getModOf6Power(e, primes1000[i]): %d\n", getModOf6Power(e, primes1000[i]));
        // printf("getModOf6Power(f, primes1000[i]): %d\n", getModOf6Power(f, primes1000[i]));
        // printf("primes1000[i]: %d\n", primes1000[i]);
        // printf("aInt: %d, bInt: %d, cInt: %d, dInt: %d, eInt: %d, fInt: %d, i: %d\n", a, b, c, d, e, f, i);
        // printf("result before while: %d\n", result);
        
        while (result < 0){
            result += primes1000[i];
        }
        if(result != 0){
            // printf("getModOf6Power(a, primes1000[i]): %d\n", getModOf6Power(a, primes1000[i]));
            // printf("getModOf6Power(b, primes1000[i]): %d\n", getModOf6Power(b, primes1000[i]));
            // printf("getModOf6Power(c, primes1000[i]): %d\n", getModOf6Power(c, primes1000[i]));
            // printf("getModOf6Power(d, primes1000[i]): %d\n", getModOf6Power(d, primes1000[i]));
            // printf("getModOf6Power(e, primes1000[i]): %d\n", getModOf6Power(e, primes1000[i]));
            // printf("getModOf6Power(f, primes1000[i]): %d\n", getModOf6Power(f, primes1000[i]));
            // printf("aInt: %d, bInt: %d, cInt: %d, dInt: %d, eInt: %d, fInt: %d, i: %d\n", a, b, c, d, e, f, i);
            // printf("primes1000[i]: %d\n", primes1000[i]);
            // printf("i in isABCDEFModEqual is :%d, result is: %d.\n", i, result);
            return false;
        }
    }

    return true;
}


__device__
bool isDecomposableIn2Numbers(int a, int b, int c, int d){
    //bool result = true;
    //int primes[12] = {7,13,19,31,37,43,61,67,73,79,109,139};
    int tmp = getLeftModSubIn2Numbers(7,a,b,c,d);
    switch(tmp){ case  4:  case  5:  case  6:
        return false;
    }
    
    tmp = getLeftModSubIn2Numbers(13,a,b,c,d);
    switch(tmp){  case 3:  case  4:  case  9:  case  10:
        return false;
    }
    tmp = getLeftModSubIn2Numbers(19,a,b,c,d);
    switch(tmp){  case 4:  case  5:  case  6:  case  9:  case  10:  case  13:  case  15:  case  16:  case  17:
        return false;
    }

    tmp = getLeftModSubIn2Numbers(31,a,b,c,d);
    switch(tmp){  case 7:  case  11:  case  13:  case  14:  case  15:  case  19: case  21:  case  22:  case  23:  case  25:  case  26:  case  27: case  28:  case  29:  case  30:
        return false;
    }
    tmp = getLeftModSubIn2Numbers(37,a,b,c,d);
    switch(tmp){  case 3:  case  4: case 5:  case  6:  case  7:  case  8:  case  13: case 14:  case  18:  case  19:  case  23:  case  24:  case  29:  case  30: case  31:  case  32:  case  33:  case  34:
        return false;
    }
    tmp = getLeftModSubIn2Numbers(43,a,b,c,d);
    switch(tmp){  case 6:  case  7:  case  10:  case  18: case  23:  case  24:  case  26: case  28:  case  29:  case  30:  case  31:  case  34:  case  38:  case  40:
        return false;
    }
    tmp = getLeftModSubIn2Numbers(61,a,b,c,d);
    switch(tmp){  case 5:  case  13:  case  15:  case  16:  case  22:  case  39: case  45:  case  46:  case  48:  case  56:
        return false;
    }

    tmp = getLeftModSubIn2Numbers(67,a,b,c,d);
    switch(tmp){  case 3:  case  5:  case  8:  case  27:  case  42:  case  43:  case  45: case  52:  case  53:  case  58:  case  66:
        return false;
    }
    tmp = getLeftModSubIn2Numbers(73,a,b,c,d);
    switch(tmp){  case 13:  case  14:  case  20:  case  29:  case  31:  case  34:  case  39: case  42:  case  44:  case  53:  case  59:  case  60:
        return false;
    }

    tmp = getLeftModSubIn2Numbers(79,a,b,c,d);
    switch(tmp){  case 12:  case  14:  case  15:  case  17:  case  27:  case  33:  case  41: case  57:  case  58:  case  61:  case  69:  case  71:  case  78:
        return false;
    }
    tmp = getLeftModSubIn2Numbers(109,a,b,c,d);
    switch(tmp){  case 6:  case  10:  case  13:  case  14:  case  24:  case  40:  case  51: case  52:  case  53:  case  56:  case  57:  case  58:  case  69:  case  85: case  95:  case  96: case  99:  case  103:
        return false;
    }
    tmp = getLeftModSubIn2Numbers(139,a,b,c,d);
    switch(tmp){  case 8:  case  10:  case  14:  case  23:  case  27:  case  33:  case  39: case  48:  case  59:  case  60:  case  62:  case  74:  case  75:  case  76: case  82:  case  84:  case  87:  case  94:  case  95:  case  103:  case  105: case  133:  case  138:
        return false;
    }
    return true;
}

__device__
bool isDecomposableIn1Numbers(int a, int b, int c, int d, int e){
    //bool result = true;
    //int primes[12] = {7,13,19,31,37,43,61,67,73,79,109,139};
    //int primes[15] = {3,5,7,11,13,17,19,23,29,31,37,41,43,47,53};
    int tmp = getLeftModSubIn1Numbers(3,a,b,c,d,e);
    switch(tmp){  case 2:
        return false;
    }
    tmp = getLeftModSubIn1Numbers(5,a,b,c,d,e);
    switch(tmp){  case 2:  case  3:
    return false;
    }
    tmp = getLeftModSubIn1Numbers(7,a,b,c,d,e);
    switch(tmp){  case 3:  case  4:  case  5:  case  6:
        return false;
    }
    tmp = getLeftModSubIn1Numbers(11,a,b,c,d,e);
    switch(tmp){  case 2:  case  6:  case  7:  case  8:  case  10:
        return false;
    }
    tmp = getLeftModSubIn1Numbers(13,a,b,c,d,e);
    switch(tmp){  case 2:  case  3:  case  4:  case  5:  case  8:  case  9:  case  10:  case  11:
        return false;
    }
    tmp = getLeftModSubIn1Numbers(17,a,b,c,d,e);
    switch(tmp){  case 3:  case  5:  case  6:  case  7:  case  10:  case  11:  case  12:  case  14:
        return false;
    }
    tmp = getLeftModSubIn1Numbers(19,a,b,c,d,e);
    switch(tmp){  case 2:  case  3:  case  4:  case  5:  case  6:  case  8:  case  9:  case  10:  case  12:  case  13:  case  14:  case  15:  case  16:  case  17:  case  18:
        return false;
    }
    tmp = getLeftModSubIn1Numbers(23,a,b,c,d,e);
    switch(tmp){  case 5:  case  7:  case  10:  case  11:  case  14:  case  15:  case  17:  case  19:  case  20:  case  21:  case  22:
        return false;
    }    
    tmp = getLeftModSubIn1Numbers(29,a,b,c,d,e);
    switch(tmp){  case 2:  case  3:  case  8:  case  10:  case  11:  case  12:  case  14:  case  15:  case  17:  case  18:  case  19:  case  21:  case  26:  case  27:
        return false;
    }
    tmp = getLeftModSubIn1Numbers(31,a,b,c,d,e);
    switch(tmp){  case 3:  case  5:  case  6:  case  7:  case  9:  case  10:  case  11:  case  12:  case  13:  case  14:  case  15:  case  17:  case  18:  case  19:  case  20:  case  21:  case  22:  case  23:  case  24:  case  25:  case  26:  case  27:  case  28:  case  29:  case  30:
        return false;
    }
    tmp = getLeftModSubIn1Numbers(37,a,b,c,d,e);
    switch(tmp){  case 2:  case  3:  case  4:  case  5:  case  6:  case  7:  case  8:  case  9:  case  12:  case  13:  case  14:  case  15:  case  16:  case  17:  case  18:  case  19:  case  20:  case  21:  case  22:  case  23:  case  24:  case  25:  case  28:  case  29:  case  30:  case  31:  case  32:  case  33:  case  34:  case  35:
        return false;
    }
    tmp = getLeftModSubIn1Numbers(41,a,b,c,d,e);
    switch(tmp){  case 3:  case  6:  case  7:  case  11:  case  12:  case  13:  case  14:  case  15:  case  17:  case  19:  case  22:  case  24:  case  26:  case  27:  case  28:  case  29:  case  30:  case  34:  case  35:  case  38:
        return false;
    }
    tmp = getLeftModSubIn1Numbers(43,a,b,c,d,e);
    switch(tmp){  case 2:  case  3:  case  5:  case  6:  case  7:  case  8:  case  9:  case  10:  case  12:  case  13:  case  14:  case  15:  case  17:  case  18:  case  19:  case  20:  case  22:  case  23:  case  24:  case  25:  case  26:  case  27:  case  28:  case  29:  case  30:  case  31:  case  32:  case  33:  case  34:  case  36:  case  37:  case  38:  case  39:  case  40:  case  42:
        return false;
    }
    tmp = getLeftModSubIn1Numbers(47,a,b,c,d,e);
    switch(tmp){  case 5:  case  10:  case  11:  case  13:  case  15:  case  19:  case  20:  case  22:  case  23:  case  26:  case  29:  case  30:  case  31:  case  33:  case  35:  case  38:  case  39:  case  40:  case  41:  case  43:  case  44:  case  45:  case  46:
        return false;
    }
    tmp = getLeftModSubIn1Numbers(53,a,b,c,d,e);
    switch(tmp){  case 2:  case  3:  case  5:  case  8:  case  12:  case  14:  case  18:  case  19:  case  20:  case  21:  case  22:  case  23:  case  26:  case  27:  case  30:  case  31:  case  32:  case  33:  case  34:  case  35:  case  39:  case  41:  case  45:  case  48:  case  50:  case  51:
        return false;
    }
    tmp = getLeftModSubIn1Numbers(59,a,b,c,d,e);
    switch(tmp){  case 2:  case  6:  case  8:  case  10:  case  11:  case  13:  case  14:  case  18:  case  23:  case  24:  case  30:  case  31:  case  32:  case  33:  case  34:  case  37:  case  38:  case  39:  case  40:  case  42:  case  43:  case  44:  case  47:  case  50:  case  52:  case  54:  case  55:  case  56:  case  58:
        return false;
    }
    tmp = getLeftModSubIn1Numbers(61,a,b,c,d,e);
    switch(tmp){  case 2:  case  4:  case  5:  case  6:  case  7:  case  8:  case  10:  case  11:  case  12:  case  13:  case  14:  case  15:  case  16:  case  17:  case  18:  case  19:  case  21:  case  22:  case  23:  case  24:  case  25:  case  26:  case  28:  case  29:  case  30:  case  31:  case  32:  case  33:  case  35:  case  36:  case  37:  case  38:  case  39:  case  40:  case  42:  case  43:  case  44:  case  45:  case  46:  case  47:  case  48:  case  49:  case  50:  case  51:  case  53:  case  54:  case  55:  case  56:  case  57:  case  59:
        return false;
    }
    tmp = getLeftModSubIn1Numbers(67,a,b,c,d,e);
    switch(tmp){  case 2:  case  3:  case  4:  case  5:  case  6:  case  7:  case  8:  case  10:  case  11:  case  12:  case  13:  case  16:  case  17:  case  18:  case  19:  case  20:  case  21:  case  23:  case  26:  case  27:  case  28:  case  29:  case  30:  case  31:  case  32:  case  33:  case  34:  case  35:  case  36:  case  37:  case  38:  case  39:  case  41:  case  42:  case  43:  case  44:  case  45:  case  46:  case  47:  case  48:  case  49:  case  50:  case  51:  case  52:  case  53:  case  54:  case  55:  case  56:  case  57:  case  58:  case  60:  case  61:  case  63:  case  65:  case  66:
        return false;
    }
    tmp = getLeftModSubIn1Numbers(71,a,b,c,d,e);
    switch(tmp){  case 7:  case  11:  case  13:  case  14:  case  17:  case  21:  case  22:  case  23:  case  26:  case  28:  case  31:  case  33:  case  34:  case  35:  case  39:  case  41:  case  42:  case  44:  case  46:  case  47:  case  51:  case  52:  case  53:  case  55:  case  56:  case  59:  case  61:  case  62:  case  63:  case  65:  case  66:  case  67:  case  68:  case  69:  case  70:
        return false;
    }
    tmp = getLeftModSubIn1Numbers(73,a,b,c,d,e);
    switch(tmp){  case 2:  case  4:  case  5:  case  6:  case  7:  case  10:  case  11:  case  12:  case  13:  case  14:  case  15:  case  16:  case  17:  case  18:  case  19:  case  20:  case  21:  case  22:  case  23:  case  25:  case  26:  case  28:  case  29:  case  30:  case  31:  case  32:  case  33:  case  34:  case  35:  case  36:  case  37:  case  38:  case  39:  case  40:  case  41:  case  42:  case  43:  case  44:  case  45:  case  47:  case  48:  case  50:  case  51:  case  52:  case  53:  case  54:  case  55:  case  56:  case  57:  case  58:  case  59:  case  60:  case  61:  case  62:  case  63:  case  66:  case  67:  case  68:  case  69:  case  71:
        return false;
    }
    tmp = getLeftModSubIn1Numbers(79,a,b,c,d,e);
    switch(tmp){  case 2:  case  3:  case  4:  case  5:  case  6:  case  7:  case  9:  case  11:  case  12:  case  13:  case  14:  case  15:  case  16:  case  17:  case  19:  case  20:  case  23:  case  24:  case  25:  case  26:  case  27:  case  28:  case  29:  case  30:  case  31:  case  32:  case  33:  case  34:  case  35:  case  36:  case  37:  case  39:  case  40:  case  41:  case  42:  case  43:  case  44:  case  45:  case  47:  case  48:  case  49:  case  50:  case  51:  case  53:  case  54:  case  55:  case  56:  case  57:  case  58:  case  59:  case  60:  case  61:  case  63:  case  66:  case  68:  case  69:  case  70:  case  71:  case  72:  case  73:  case  74:  case  75:  case  76:  case  77:  case  78:
        return false;
    }
    tmp = getLeftModSubIn1Numbers(83,a,b,c,d,e);
    switch(tmp){  case 2:  case  5:  case  6:  case  8:  case  13:  case  14:  case  15:  case  18:  case  19:  case  20:  case  22:  case  24:  case  32:  case  34:  case  35:  case  39:  case  42:  case  43:  case  45:  case  46:  case  47:  case  50:  case  52:  case  53:  case  54:  case  55:  case  56:  case  57:  case  58:  case  60:  case  62:  case  66:  case  67:  case  71:  case  72:  case  73:  case  74:  case  76:  case  79:  case  80:  case  82:
        return false;
    }
    tmp = getLeftModSubIn1Numbers(89,a,b,c,d,e);
    switch(tmp){  case 3:  case  6:  case  7:  case  12:  case  13:  case  14:  case  15:  case  19:  case  23:  case  24:  case  26:  case  27:  case  28:  case  29:  case  30:  case  31:  case  33:  case  35:  case  37:  case  38:  case  41:  case  43:  case  46:  case  48:  case  51:  case  52:  case  54:  case  56:  case  58:  case  59:  case  60:  case  61:  case  62:  case  63:  case  65:  case  66:  case  70:  case  74:  case  75:  case  76:  case  77:  case  82:  case  83:  case  86:
        return false;
    }
    tmp = getLeftModSubIn1Numbers(97,a,b,c,d,e);
    switch(tmp){  case 2:  case  3:  case  4:  case  5:  case  6:  case  7:  case  9:  case  10:  case  11:  case  13:  case  14:  case  15:  case  16:  case  17:  case  19:  case  20:  case  21:  case  23:  case  24:  case  25:  case  26:  case  28:  case  29:  case  30:  case  31:  case  32:  case  34:  case  35:  case  36:  case  37:  case  38:  case  39:  case  40:  case  41:  case  42:  case  43:  case  44:  case  45:  case  46:  case  48:  case  49:  case  51:  case  52:  case  53:  case  54:  case  55:  case  56:  case  57:  case  58:  case  59:  case  60:  case  61:  case  62:  case  63:  case  65:  case  66:  case  67:  case  68:  case  69:  case  71:  case  72:  case  73:  case  74:  case  76:  case  77:  case  78:  case  80:  case  81:  case  82:  case  83:  case  84:  case  86:  case  87:  case  88:  case  90:  case  91:  case  92:  case  93:  case  94:  case  95:
        return false;
    }
    
    
    tmp = getLeftModSubIn1Numbers(101,a,b,c,d,e);
    switch(tmp){  case 2:  case  3:  case  7:  case  8:  case  10:  case  11:  case  12:  case  15:  case  18:  case  26:  case  27:  case  28:  case  29:  case  32:  case  34:  case  35:  case  38:  case  39:  case  40:  case  41:  case  42:  case  44:  case  46:  case  48:  case  50:  case  51:  case  53:  case  55:  case  57:  case  59:  case  60:  case  61:  case  62:  case  63:  case  66:  case  67:  case  69:  case  72:  case  73:  case  74:  case  75:  case  83:  case  86:  case  89:  case  90:  case  91:  case  93:  case  94:  case  98:  case  99:
    return false;}
    tmp = getLeftModSubIn1Numbers(103,a,b,c,d,e);
    switch(tmp){  case 2:  case  3:  case  4:  case  5:  case  6:  case  7:  case  10:  case  11:  case  12:  case  15:  case  16:  case  17:  case  18:  case  19:  case  20:  case  21:  case  22:  case  24:  case  25:  case  26:  case  27:  case  28:  case  29:  case  31:  case  32:  case  33:  case  35:  case  36:  case  37:  case  38:  case  39:  case  40:  case  41:  case  42:  case  43:  case  44:  case  45:  case  46:  case  47:  case  48:  case  49:  case  50:  case  51:  case  52:  case  53:  case  54:  case  55:  case  56:  case  57:  case  58:  case  59:  case  60:  case  62:  case  63:  case  65:  case  67:  case  68:  case  69:  case  70:  case  71:  case  73:  case  74:  case  75:  case  77:  case  78:  case  80:  case  82:  case  83:  case  84:  case  85:  case  86:  case  87:  case  88:  case  89:  case  90:  case  91:  case  92:  case  94:  case  95:  case  96:  case  97:  case  98:  case  99:  case  101:  case  102:
    return false;}
    tmp = getLeftModSubIn1Numbers(107,a,b,c,d,e);
    switch(tmp){  case 2:  case  5:  case  6:  case  7:  case  8:  case  15:  case  17:  case  18:  case  20:  case  21:  case  22:  case  24:  case  26:  case  28:  case  31:  case  32:  case  38:  case  43:  case  45:  case  46:  case  50:  case  51:  case  54:  case  55:  case  58:  case  59:  case  60:  case  63:  case  65:  case  66:  case  67:  case  68:  case  70:  case  71:  case  72:  case  73:  case  74:  case  77:  case  78:  case  80:  case  82:  case  84:  case  88:  case  91:  case  93:  case  94:  case  95:  case  96:  case  97:  case  98:  case  103:  case  104:  case  106:
    return false;}
    tmp = getLeftModSubIn1Numbers(109,a,b,c,d,e);
    switch(tmp){  case 2:  case  3:  case  5:  case  6:  case  7:  case  8:  case  9:  case  10:  case  11:  case  12:  case  13:  case  14:  case  15:  case  17:  case  18:  case  19:  case  20:  case  21:  case  22:  case  23:  case  24:  case  25:  case  26:  case  28:  case  29:  case  30:  case  31:  case  32:  case  33:  case  35:  case  36:  case  37:  case  39:  case  40:  case  41:  case  42:  case  44:  case  47:  case  48:  case  49:  case  50:  case  51:  case  52:  case  53:  case  54:  case  55:  case  56:  case  57:  case  58:  case  59:  case  60:  case  61:  case  62:  case  65:  case  67:  case  68:  case  69:  case  70:  case  72:  case  73:  case  74:  case  76:  case  77:  case  78:  case  79:  case  80:  case  81:  case  83:  case  84:  case  85:  case  86:  case  87:  case  88:  case  89:  case  90:  case  91:  case  92:  case  94:  case  95:  case  96:  case  97:  case  98:  case  99:  case  100:  case  101:  case  102:  case  103:  case  104:  case  106:  case  107:
    return false;}
    tmp = getLeftModSubIn1Numbers(113,a,b,c,d,e);
    switch(tmp){  case 3:  case  5:  case  6:  case  10:  case  12:  case  17:  case  19:  case  20:  case  21:  case  23:  case  24:  case  27:  case  29:  case  33:  case  34:  case  35:  case  37:  case  38:  case  39:  case  40:  case  42:  case  43:  case  45:  case  46:  case  47:  case  48:  case  54:  case  55:  case  58:  case  59:  case  65:  case  66:  case  67:  case  68:  case  70:  case  71:  case  73:  case  74:  case  75:  case  76:  case  78:  case  79:  case  80:  case  84:  case  86:  case  89:  case  90:  case  92:  case  93:  case  94:  case  96:  case  101:  case  103:  case  107:  case  108:  case  110:
    return false;}
    tmp = getLeftModSubIn1Numbers(127,a,b,c,d,e);
    switch(tmp){  case 3:  case  5:  case  6:  case  7:  case  9:  case  10:  case  11:  case  12:  case  13:  case  14:  case  15:  case  17:  case  18:  case  20:  case  21:  case  22:  case  23:  case  24:  case  26:  case  27:  case  28:  case  29:  case  30:  case  31:  case  33:  case  34:  case  35:  case  36:  case  37:  case  39:  case  40:  case  41:  case  42:  case  43:  case  44:  case  45:  case  46:  case  48:  case  49:  case  51:  case  52:  case  53:  case  54:  case  55:  case  56:  case  57:  case  58:  case  59:  case  60:  case  62:  case  63:  case  65:  case  66:  case  67:  case  68:  case  69:  case  70:  case  71:  case  72:  case  74:  case  75:  case  77:  case  78:  case  79:  case  80:  case  81:  case  82:  case  83:  case  84:  case  85:  case  86:  case  88:  case  89:  case  90:  case  91:  case  92:  case  93:  case  95:  case  96:  case  97:  case  98:  case  99:  case  101:  case  102:  case  103:  case  104:  case  105:  case  106:  case  108:  case  109:  case  110:  case  111:  case  112:  case  113:  case  114:  case  115:  case  116:  case  118:  case  119:  case  120:  case  121:  case  123:  case  124:  case  125:  case  126:
    return false;}
    tmp = getLeftModSubIn1Numbers(131,a,b,c,d,e);
    switch(tmp){  case 2:  case  6:  case  8:  case  10:  case  14:  case  17:  case  18:  case  19:  case  22:  case  23:  case  24:  case  26:  case  29:  case  30:  case  31:  case  32:  case  37:  case  40:  case  42:  case  47:  case  50:  case  51:  case  54:  case  56:  case  57:  case  66:  case  67:  case  68:  case  69:  case  70:  case  71:  case  72:  case  73:  case  76:  case  78:  case  79:  case  82:  case  83:  case  85:  case  86:  case  87:  case  88:  case  90:  case  92:  case  93:  case  95:  case  96:  case  97:  case  98:  case  103:  case  104:  case  106:  case  110:  case  111:  case  115:  case  116:  case  118:  case  119:  case  120:  case  122:  case  124:  case  126:  case  127:  case  128:  case  130:
    return false;}
    tmp = getLeftModSubIn1Numbers(137,a,b,c,d,e);
    switch(tmp){  case 3:  case  5:  case  6:  case  10:  case  12:  case  13:  case  20:  case  21:  case  23:  case  24:  case  26:  case  27:  case  29:  case  31:  case  33:  case  35:  case  40:  case  41:  case  42:  case  43:  case  45:  case  46:  case  47:  case  48:  case  51:  case  52:  case  53:  case  54:  case  55:  case  57:  case  58:  case  62:  case  66:  case  67:  case  70:  case  71:  case  75:  case  79:  case  80:  case  82:  case  83:  case  84:  case  85:  case  86:  case  89:  case  90:  case  91:  case  92:  case  94:  case  95:  case  96:  case  97:  case  102:  case  104:  case  106:  case  108:  case  110:  case  111:  case  113:  case  114:  case  116:  case  117:  case  124:  case  125:  case  127:  case  131:  case  132:  case  134:
    return false;}
    tmp = getLeftModSubIn1Numbers(139,a,b,c,d,e);
    switch(tmp){  case 2:  case  3:  case  4:  case  5:  case  7:  case  8:  case  9:  case  10:  case  11:  case  12:  case  13:  case  14:  case  15:  case  16:  case  17:  case  18:  case  19:  case  20:  case  21:  case  22:  case  23:  case  24:  case  25:  case  26:  case  27:  case  28:  case  29:  case  30:  case  31:  case  32:  case  33:  case  35:  case  37:  case  38:  case  39:  case  40:  case  41:  case  42:  case  43:  case  46:  case  47:  case  48:  case  49:  case  50:  case  51:  case  53:  case  54:  case  56:  case  58:  case  59:  case  60:  case  61:  case  62:  case  66:  case  67:  case  68:  case  69:  case  70:  case  71:  case  72:  case  73:  case  74:  case  75:  case  76:  case  78:  case  81:  case  82:  case  83:  case  84:  case  85:  case  86:  case  87:  case  88:  case  89:  case  90:  case  92:  case  93:  case  94:  case  95:  case  96:  case  97:  case  98:  case  99:  case  101:  case  102:  case  103:  case  104:  case  105:  case  107:  case  108:  case  109:  case  110:  case  111:  case  113:  case  114:  case  115:  case  117:  case  118:  case  119:  case  120:  case  121:  case  122:  case  123:  case  124:  case  126:  case  127:  case  128:  case  130:  case  132:  case  133:  case  134:  case  135:  case  136:  case  137:  case  138:
    return false;}
    tmp = getLeftModSubIn1Numbers(149,a,b,c,d,e);
    switch(tmp){  case 2:  case  3:  case  8:  case  10:  case  11:  case  12:  case  13:  case  14:  case  15:  case  18:  case  21:  case  23:  case  27:  case  32:  case  34:  case  38:  case  40:  case  41:  case  43:  case  44:  case  48:  case  50:  case  51:  case  52:  case  55:  case  56:  case  57:  case  58:  case  59:  case  60:  case  62:  case  65:  case  66:  case  70:  case  71:  case  72:  case  74:  case  75:  case  77:  case  78:  case  79:  case  83:  case  84:  case  87:  case  89:  case  90:  case  91:  case  92:  case  93:  case  94:  case  97:  case  98:  case  99:  case  101:  case  105:  case  106:  case  108:  case  109:  case  111:  case  115:  case  117:  case  122:  case  126:  case  128:  case  131:  case  134:  case  135:  case  136:  case  137:  case  138:  case  139:  case  141:  case  146:  case  147:
    return false;}
    tmp = getLeftModSubIn1Numbers(151,a,b,c,d,e);
    switch(tmp){  case 2:  case  3:  case  4:  case  5:  case  6:  case  7:  case  10:  case  11:  case  12:  case  13:  case  14:  case  15:  case  16:  case  17:  case  18:  case  21:  case  22:  case  23:  case  24:  case  25:  case  26:  case  27:  case  28:  case  30:  case  31:  case  32:  case  33:  case  34:  case  35:  case  36:  case  37:  case  38:  case  39:  case  40:  case  41:  case  42:  case  43:  case  45:  case  46:  case  47:  case  48:  case  49:  case  51:  case  52:  case  53:  case  54:  case  55:  case  56:  case  57:  case  58:  case  60:  case  61:  case  62:  case  63:  case  65:  case  66:  case  67:  case  69:  case  70:  case  71:  case  73:  case  74:  case  75:  case  76:  case  77:  case  79:  case  80:  case  82:  case  83:  case  85:  case  87:  case  88:  case  89:  case  90:  case  92:  case  93:  case  95:  case  96:  case  97:  case  99:  case  100:  case  101:  case  102:  case  103:  case  104:  case  105:  case  106:  case  107:  case  108:  case  109:  case  111:  case  112:  case  113:  case  114:  case  115:  case  116:  case  117:  case  118:  case  119:  case  120:  case  121:  case  122:  case  126:  case  128:  case  129:  case  130:  case  131:  case  132:  case  133:  case  134:  case  135:  case  136:  case  137:  case  138:  case  139:  case  140:  case  141:  case  142:  case  143:  case  144:  case  145:  case  146:  case  147:  case  149:  case  150:
    return false;}
    tmp = getLeftModSubIn1Numbers(157,a,b,c,d,e);
    switch(tmp){  case 2:  case  3:  case  5:  case  6:  case  7:  case  8:  case  9:  case  10:  case  11:  case  12:  case  13:  case  15:  case  17:  case  18:  case  19:  case  20:  case  21:  case  22:  case  23:  case  24:  case  25:  case  26:  case  28:  case  29:  case  30:  case  31:  case  32:  case  33:  case  34:  case  35:  case  36:  case  37:  case  38:  case  40:  case  41:  case  42:  case  43:  case  44:  case  45:  case  47:  case  48:  case  50:  case  51:  case  52:  case  53:  case  54:  case  55:  case  57:  case  59:  case  60:  case  61:  case  62:  case  63:  case  65:  case  66:  case  68:  case  69:  case  70:  case  71:  case  72:  case  73:  case  74:  case  76:  case  77:  case  78:  case  79:  case  80:  case  81:  case  83:  case  84:  case  85:  case  86:  case  87:  case  88:  case  89:  case  91:  case  92:  case  94:  case  95:  case  96:  case  97:  case  98:  case  100:  case  102:  case  103:  case  104:  case  105:  case  106:  case  107:  case  109:  case  110:  case  112:  case  113:  case  114:  case  115:  case  116:  case  117:  case  119:  case  120:  case  121:  case  122:  case  123:  case  124:  case  125:  case  126:  case  127:  case  128:  case  129:  case  131:  case  132:  case  133:  case  134:  case  135:  case  136:  case  137:  case  138:  case  139:  case  140:  case  142:  case  144:  case  145:  case  146:  case  147:  case  148:  case  149:  case  150:  case  151:  case  152:  case  154:  case  155:
    return false;}
    tmp = getLeftModSubIn1Numbers(163,a,b,c,d,e);
    switch(tmp){  case 2:  case  3:  case  4:  case  5:  case  7:  case  8:  case  9:  case  10:  case  11:  case  12:  case  13:  case  14:  case  15:  case  16:  case  17:  case  18:  case  19:  case  20:  case  23:  case  24:  case  26:  case  27:  case  28:  case  29:  case  30:  case  31:  case  32:  case  33:  case  34:  case  35:  case  37:  case  39:  case  41:  case  42:  case  43:  case  44:  case  45:  case  46:  case  47:  case  48:  case  49:  case  50:  case  51:  case  52:  case  54:  case  55:  case  56:  case  57:  case  59:  case  60:  case  62:  case  63:  case  66:  case  67:  case  68:  case  69:  case  70:  case  71:  case  72:  case  73:  case  74:  case  75:  case  76:  case  78:  case  79:  case  80:  case  81:  case  82:  case  83:  case  84:  case  86:  case  87:  case  88:  case  89:  case  90:  case  91:  case  92:  case  93:  case  94:  case  95:  case  96:  case  97:  case  98:  case  99:  case  100:  case  101:  case  102:  case  103:  case  105:  case  106:  case  107:  case  108:  case  109:  case  110:  case  111:  case  112:  case  113:  case  114:  case  116:  case  117:  case  118:  case  119:  case  120:  case  121:  case  122:  case  123:  case  124:  case  125:  case  127:  case  128:  case  129:  case  130:  case  131:  case  134:  case  137:  case  138:  case  139:  case  141:  case  142:  case  143:  case  144:  case  145:  case  147:  case  148:  case  149:  case  151:  case  152:  case  153:  case  154:  case  156:  case  157:  case  159:  case  160:  case  161:  case  162:
    return false;}
    tmp = getLeftModSubIn1Numbers(167,a,b,c,d,e);
    switch(tmp){  case 5:  case  10:  case  13:  case  15:  case  17:  case  20:  case  23:  case  26:  case  30:  case  34:  case  35:  case  37:  case  39:  case  40:  case  41:  case  43:  case  45:  case  46:  case  51:  case  52:  case  53:  case  55:  case  59:  case  60:  case  67:  case  68:  case  69:  case  70:  case  71:  case  73:  case  74:  case  78:  case  79:  case  80:  case  82:  case  83:  case  86:  case  90:  case  91:  case  92:  case  95:  case  101:  case  102:  case  103:  case  104:  case  105:  case  106:  case  109:  case  110:  case  111:  case  113:  case  117:  case  118:  case  119:  case  120:  case  123:  case  125:  case  129:  case  131:  case  134:  case  135:  case  136:  case  138:  case  139:  case  140:  case  142:  case  143:  case  145:  case  146:  case  148:  case  149:  case  151:  case  153:  case  155:  case  156:  case  158:  case  159:  case  160:  case  161:  case  163:  case  164:  case  165:  case  166:
    return false;}
    tmp = getLeftModSubIn1Numbers(173,a,b,c,d,e);
    switch(tmp){  case 2:  case  3:  case  5:  case  7:  case  8:  case  11:  case  12:  case  17:  case  18:  case  19:  case  20:  case  26:  case  27:  case  28:  case  30:  case  32:  case  39:  case  42:  case  44:  case  45:  case  46:  case  48:  case  50:  case  53:  case  58:  case  59:  case  61:  case  62:  case  63:  case  65:  case  66:  case  68:  case  69:  case  70:  case  71:  case  72:  case  74:  case  75:  case  76:  case  79:  case  80:  case  82:  case  86:  case  87:  case  91:  case  93:  case  94:  case  97:  case  98:  case  99:  case  101:  case  102:  case  103:  case  104:  case  105:  case  107:  case  108:  case  110:  case  111:  case  112:  case  114:  case  115:  case  120:  case  123:  case  125:  case  127:  case  128:  case  129:  case  131:  case  134:  case  141:  case  143:  case  145:  case  146:  case  147:  case  153:  case  154:  case  155:  case  156:  case  161:  case  162:  case  165:  case  166:  case  168:  case  170:  case  171:
    return false;}
    tmp = getLeftModSubIn1Numbers(179,a,b,c,d,e);
    switch(tmp){  case 2:  case  6:  case  7:  case  8:  case  10:  case  11:  case  18:  case  21:  case  23:  case  24:  case  26:  case  28:  case  30:  case  32:  case  33:  case  34:  case  35:  case  37:  case  38:  case  40:  case  41:  case  44:  case  50:  case  53:  case  54:  case  55:  case  58:  case  62:  case  63:  case  69:  case  71:  case  72:  case  73:  case  78:  case  79:  case  84:  case  86:  case  90:  case  91:  case  92:  case  94:  case  96:  case  97:  case  98:  case  99:  case  102:  case  103:  case  104:  case  105:  case  109:  case  111:  case  112:  case  113:  case  114:  case  115:  case  118:  case  119:  case  120:  case  122:  case  123:  case  127:  case  128:  case  130:  case  131:  case  132:  case  133:  case  134:  case  136:  case  137:  case  140:  case  143:  case  148:  case  150:  case  152:  case  154:  case  157:  case  159:  case  160:  case  162:  case  163:  case  164:  case  165:  case  166:  case  167:  case  170:  case  174:  case  175:  case  176:  case  178:
    return false;}
    tmp = getLeftModSubIn1Numbers(181,a,b,c,d,e);
    switch(tmp){  case 2:  case  3:  case  4:  case  6:  case  7:  case  8:  case  9:  case  10:  case  11:  case  12:  case  13:  case  14:  case  15:  case  16:  case  17:  case  18:  case  19:  case  20:  case  21:  case  22:  case  23:  case  24:  case  26:  case  28:  case  30:  case  31:  case  32:  case  33:  case  34:  case  35:  case  37:  case  38:  case  39:  case  40:  case  41:  case  43:  case  44:  case  45:  case  47:  case  50:  case  51:  case  52:  case  53:  case  54:  case  55:  case  57:  case  58:  case  60:  case  61:  case  62:  case  63:  case  65:  case  66:  case  68:  case  69:  case  70:  case  71:  case  72:  case  73:  case  74:  case  75:  case  76:  case  77:  case  78:  case  79:  case  80:  case  81:  case  83:  case  84:  case  85:  case  86:  case  87:  case  88:  case  89:  case  90:  case  91:  case  92:  case  93:  case  94:  case  95:  case  96:  case  97:  case  98:  case  100:  case  101:  case  102:  case  103:  case  104:  case  105:  case  106:  case  107:  case  108:  case  109:  case  110:  case  111:  case  112:  case  113:  case  115:  case  116:  case  118:  case  119:  case  120:  case  121:  case  123:  case  124:  case  126:  case  127:  case  128:  case  129:  case  130:  case  131:  case  134:  case  136:  case  137:  case  138:  case  140:  case  141:  case  142:  case  143:  case  144:  case  146:  case  147:  case  148:  case  149:  case  150:  case  151:  case  153:  case  155:  case  157:  case  158:  case  159:  case  160:  case  161:  case  162:  case  163:  case  164:  case  165:  case  166:  case  167:  case  168:  case  169:  case  170:  case  171:  case  172:  case  173:  case  174:  case  175:  case  177:  case  178:  case  179:
    return false;}
    tmp = getLeftModSubIn1Numbers(191,a,b,c,d,e);
    switch(tmp){  case 7:  case  11:  case  14:  case  19:  case  21:  case  22:  case  28:  case  29:  case  31:  case  33:  case  35:  case  37:  case  38:  case  41:  case  42:  case  44:  case  47:  case  53:  case  55:  case  56:  case  57:  case  58:  case  61:  case  62:  case  63:  case  66:  case  70:  case  71:  case  73:  case  74:  case  76:  case  82:  case  83:  case  84:  case  87:  case  88:  case  89:  case  91:  case  93:  case  94:  case  95:  case  99:  case  101:  case  105:  case  106:  case  110:  case  111:  case  112:  case  113:  case  114:  case  116:  case  119:  case  122:  case  123:  case  124:  case  126:  case  127:  case  131:  case  132:  case  137:  case  139:  case  140:  case  141:  case  142:  case  143:  case  145:  case  146:  case  148:  case  151:  case  152:  case  155:  case  157:  case  159:  case  161:  case  164:  case  165:  case  166:  case  167:  case  168:  case  171:  case  173:  case  174:  case  175:  case  176:  case  178:  case  179:  case  181:  case  182:  case  183:  case  185:  case  186:  case  187:  case  188:  case  189:  case  190:
    return false;}
    tmp = getLeftModSubIn1Numbers(193,a,b,c,d,e);
    switch(tmp){  case 2:  case  4:  case  5:  case  6:  case  7:  case  10:  case  11:  case  12:  case  13:  case  15:  case  16:  case  17:  case  18:  case  19:  case  20:  case  21:  case  22:  case  25:  case  26:  case  28:  case  29:  case  30:  case  31:  case  32:  case  33:  case  34:  case  35:  case  36:  case  37:  case  38:  case  39:  case  40:  case  41:  case  44:  case  45:  case  46:  case  47:  case  48:  case  49:  case  51:  case  52:  case  53:  case  54:  case  55:  case  56:  case  57:  case  58:  case  59:  case  60:  case  61:  case  62:  case  63:  case  65:  case  66:  case  68:  case  70:  case  71:  case  73:  case  74:  case  75:  case  76:  case  77:  case  78:  case  79:  case  80:  case  82:  case  83:  case  84:  case  85:  case  86:  case  87:  case  88:  case  89:  case  90:  case  91:  case  92:  case  93:  case  94:  case  95:  case  96:  case  97:  case  98:  case  99:  case  100:  case  101:  case  102:  case  103:  case  104:  case  105:  case  106:  case  107:  case  108:  case  109:  case  110:  case  111:  case  113:  case  114:  case  115:  case  116:  case  117:  case  118:  case  119:  case  120:  case  122:  case  123:  case  125:  case  127:  case  128:  case  130:  case  131:  case  132:  case  133:  case  134:  case  135:  case  136:  case  137:  case  138:  case  139:  case  140:  case  141:  case  142:  case  144:  case  145:  case  146:  case  147:  case  148:  case  149:  case  152:  case  153:  case  154:  case  155:  case  156:  case  157:  case  158:  case  159:  case  160:  case  161:  case  162:  case  163:  case  164:  case  165:  case  167:  case  168:  case  171:  case  172:  case  173:  case  174:  case  175:  case  176:  case  177:  case  178:  case  180:  case  181:  case  182:  case  183:  case  186:  case  187:  case  188:  case  189:  case  191:
    return false;}
    tmp = getLeftModSubIn1Numbers(197,a,b,c,d,e);
    switch(tmp){  case 2:  case  3:  case  5:  case  8:  case  11:  case  12:  case  13:  case  14:  case  17:  case  18:  case  20:  case  21:  case  27:  case  30:  case  31:  case  32:  case  35:  case  38:  case  44:  case  45:  case  46:  case  48:  case  50:  case  52:  case  56:  case  57:  case  58:  case  66:  case  67:  case  68:  case  69:  case  71:  case  72:  case  73:  case  74:  case  75:  case  77:  case  78:  case  79:  case  80:  case  82:  case  84:  case  86:  case  87:  case  89:  case  91:  case  94:  case  95:  case  98:  case  99:  case  102:  case  103:  case  106:  case  108:  case  110:  case  111:  case  113:  case  115:  case  117:  case  118:  case  119:  case  120:  case  122:  case  123:  case  124:  case  125:  case  126:  case  128:  case  129:  case  130:  case  131:  case  139:  case  140:  case  141:  case  145:  case  147:  case  149:  case  151:  case  152:  case  153:  case  159:  case  162:  case  165:  case  166:  case  167:  case  170:  case  176:  case  177:  case  179:  case  180:  case  183:  case  184:  case  185:  case  186:  case  189:  case  192:  case  194:  case  195:
    return false;}
    tmp = getLeftModSubIn1Numbers(199,a,b,c,d,e);
    switch(tmp){  case 2:  case  3:  case  4:  case  6:  case  7:  case  9:  case  10:  case  11:  case  12:  case  13:  case  14:  case  15:  case  16:  case  17:  case  19:  case  20:  case  21:  case  22:  case  23:  case  24:  case  26:  case  27:  case  29:  case  30:  case  31:  case  32:  case  33:  case  34:  case  35:  case  36:  case  37:  case  38:  case  39:  case  41:  case  42:  case  43:  case  44:  case  45:  case  46:  case  47:  case  48:  case  49:  case  50:  case  51:  case  53:  case  54:  case  55:  case  56:  case  57:  case  58:  case  59:  case  60:  case  65:  case  66:  case  67:  case  68:  case  69:  case  70:  case  71:  case  72:  case  73:  case  74:  case  75:  case  76:  case  77:  case  78:  case  79:  case  80:  case  81:  case  82:  case  83:  case  84:  case  85:  case  86:  case  87:  case  88:  case  89:  case  91:  case  93:  case  94:  case  95:  case  96:  case  97:  case  99:  case  100:  case  101:  case  102:  case  104:  case  105:  case  107:  case  108:  case  109:  case  110:  case  112:  case  113:  case  115:  case  118:  case  119:  case  120:  case  122:  case  124:  case  126:  case  127:  case  128:  case  129:  case  130:  case  131:  case  133:  case  134:  case  135:  case  136:  case  137:  case  138:  case  141:  case  142:  case  143:  case  145:  case  146:  case  147:  case  148:  case  149:  case  150:  case  151:  case  152:  case  153:  case  154:  case  155:  case  156:  case  158:  case  159:  case  160:  case  161:  case  162:  case  163:  case  164:  case  165:  case  166:  case  167:  case  168:  case  169:  case  170:  case  171:  case  173:  case  174:  case  175:  case  176:  case  177:  case  178:  case  179:  case  180:  case  181:  case  183:  case  184:  case  185:  case  186:  case  189:  case  190:  case  191:  case  192:  case  193:  case  194:  case  195:  case  196:  case  197:  case  198:
    return false;}
    


    
    tmp = getLeftModSubIn1Numbers(211,a,b,c,d,e);
    switch(tmp){  case 2:  case  3:  case  4:  case  6:  case  7:  case  8:  case  9:  case  10:  case  12:  case  14:  case  15:  case  16:  case  17:  case  18:  case  19:  case  20:  case  21:  case  22:  case  23:  case  24:  case  26:  case  27:  case  28:  case  29:  case  30:  case  31:  case  32:  case  33:  case  34:  case  35:  case  36:  case  37:  case  38:  case  39:  case  40:  case  41:  case  42:  case  43:  case  44:  case  45:  case  46:  case  47:  case  48:  case  49:  case  50:  case  51:  case  52:  case  53:  case  54:  case  56:  case  57:  case  59:  case  60:  case  61:  case  62:  case  63:  case  66:  case  67:  case  68:  case  69:  case  70:  case  72:  case  73:  case  74:  case  75:  case  77:  case  78:  case  80:  case  81:  case  83:  case  84:  case  85:  case  86:  case  88:  case  89:  case  90:  case  91:  case  92:  case  93:  case  94:  case  95:  case  97:  case  98:  case  99:  case  100:  case  101:  case  102:  case  103:  case  104:  case  105:  case  106:  case  108:  case  110:  case  111:  case  112:  case  115:  case  116:  case  117:  case  118:  case  119:  case  120:  case  124:  case  126:  case  127:  case  128:  case  129:  case  130:  case  131:  case  132:  case  133:  case  134:  case  135:  case  136:  case  137:  case  138:  case  139:  case  140:  case  141:  case  142:  case  145:  case  146:  case  147:  case  149:  case  150:  case  152:  case  153:  case  154:  case  155:  case  156:  case  157:  case  158:  case  159:  case  160:  case  161:  case  162:  case  163:  case  164:  case  165:  case  166:  case  167:  case  168:  case  170:  case  172:  case  173:  case  174:  case  175:  case  176:  case  177:  case  178:  case  179:  case  180:  case  181:  case  182:  case  185:  case  186:  case  187:  case  189:  case  190:  case  191:  case  192:  case  194:  case  195:  case  196:  case  197:  case  198:  case  200:  case  201:  case  202:  case  204:  case  205:  case  206:  case  207:  case  208:  case  209:  case  210:
    return false;}
    tmp = getLeftModSubIn1Numbers(223,a,b,c,d,e);
    switch(tmp){  case 3:  case  5:  case  6:  case  9:  case  10:  case  11:  case  12:  case  13:  case  18:  case  19:  case  20:  case  21:  case  22:  case  23:  case  24:  case  25:  case  26:  case  27:  case  29:  case  31:  case  35:  case  36:  case  37:  case  38:  case  39:  case  40:  case  42:  case  43:  case  44:  case  45:  case  46:  case  47:  case  48:  case  50:  case  51:  case  52:  case  53:  case  54:  case  55:  case  57:  case  58:  case  59:  case  61:  case  62:  case  63:  case  65:  case  67:  case  69:  case  70:  case  71:  case  72:  case  73:  case  74:  case  75:  case  76:  case  77:  case  78:  case  79:  case  80:  case  81:  case  83:  case  84:  case  85:  case  86:  case  87:  case  88:  case  89:  case  90:  case  91:  case  92:  case  93:  case  94:  case  95:  case  96:  case  97:  case  99:  case  100:  case  101:  case  102:  case  103:  case  104:  case  106:  case  107:  case  108:  case  109:  case  110:  case  111:  case  113:  case  114:  case  116:  case  117:  case  118:  case  121:  case  122:  case  123:  case  124:  case  125:  case  126:  case  127:  case  129:  case  130:  case  131:  case  133:  case  134:  case  135:  case  137:  case  138:  case  139:  case  140:  case  141:  case  142:  case  143:  case  144:  case  145:  case  146:  case  147:  case  148:  case  149:  case  150:  case  151:  case  152:  case  153:  case  154:  case  155:  case  156:  case  157:  case  158:  case  159:  case  160:  case  161:  case  162:  case  163:  case  165:  case  166:  case  167:  case  168:  case  170:  case  172:  case  173:  case  174:  case  175:  case  176:  case  177:  case  178:  case  179:  case  180:  case  181:  case  182:  case  183:  case  184:  case  185:  case  186:  case  187:  case  188:  case  189:  case  190:  case  191:  case  192:  case  193:  case  194:  case  195:  case  198:  case  199:  case  200:  case  201:  case  202:  case  203:  case  204:  case  205:  case  206:  case  207:  case  208:  case  209:  case  211:  case  212:  case  213:  case  214:  case  215:  case  216:  case  217:  case  218:  case  219:  case  220:  case  221:  case  222:
    return false;}
    tmp = getLeftModSubIn1Numbers(227,a,b,c,d,e);
    switch(tmp){  case 2:  case  5:  case  6:  case  8:  case  13:  case  14:  case  15:  case  17:  case  18:  case  20:  case  22:  case  24:  case  31:  case  32:  case  35:  case  37:  case  38:  case  39:  case  41:  case  42:  case  45:  case  46:  case  50:  case  51:  case  52:  case  54:  case  55:  case  56:  case  58:  case  60:  case  61:  case  66:  case  67:  case  68:  case  72:  case  80:  case  83:  case  86:  case  88:  case  91:  case  93:  case  94:  case  95:  case  96:  case  98:  case  105:  case  106:  case  107:  case  111:  case  114:  case  115:  case  117:  case  118:  case  119:  case  123:  case  124:  case  125:  case  126:  case  127:  case  128:  case  130:  case  135:  case  137:  case  138:  case  140:  case  142:  case  143:  case  145:  case  146:  case  148:  case  149:  case  150:  case  151:  case  152:  case  153:  case  154:  case  156:  case  157:  case  158:  case  162:  case  163:  case  164:  case  165:  case  168:  case  170:  case  174:  case  178:  case  179:  case  180:  case  183:  case  184:  case  187:  case  191:  case  193:  case  194:  case  197:  case  198:  case  199:  case  200:  case  201:  case  202:  case  204:  case  206:  case  208:  case  211:  case  215:  case  216:  case  217:  case  218:  case  220:  case  223:  case  224:  case  226:
    return false;}
    tmp = getLeftModSubIn1Numbers(229,a,b,c,d,e);
    switch(tmp){  case 2:  case  3:  case  5:  case  6:  case  7:  case  8:  case  9:  case  10:  case  12:  case  13:  case  14:  case  18:  case  19:  case  20:  case  21:  case  22:  case  23:  case  24:  case  25:  case  28:  case  29:  case  30:  case  31:  case  32:  case  33:  case  34:  case  35:  case  36:  case  37:  case  38:  case  39:  case  40:  case  41:  case  45:  case  46:  case  47:  case  48:  case  49:  case  50:  case  51:  case  52:  case  54:  case  55:  case  56:  case  58:  case  59:  case  62:  case  63:  case  65:  case  66:  case  67:  case  69:  case  70:  case  71:  case  72:  case  73:  case  74:  case  75:  case  76:  case  77:  case  78:  case  79:  case  80:  case  81:  case  82:  case  83:  case  84:  case  85:  case  86:  case  87:  case  88:  case  89:  case  90:  case  91:  case  92:  case  93:  case  94:  case  95:  case  96:  case  97:  case  98:  case  99:  case  100:  case  101:  case  102:  case  103:  case  105:  case  106:  case  107:  case  109:  case  110:  case  111:  case  112:  case  113:  case  114:  case  115:  case  116:  case  117:  case  118:  case  119:  case  120:  case  122:  case  123:  case  124:  case  126:  case  127:  case  128:  case  129:  case  130:  case  131:  case  132:  case  133:  case  134:  case  135:  case  136:  case  137:  case  138:  case  139:  case  140:  case  141:  case  142:  case  143:  case  144:  case  145:  case  146:  case  147:  case  148:  case  149:  case  150:  case  151:  case  152:  case  153:  case  154:  case  155:  case  156:  case  157:  case  158:  case  159:  case  160:  case  162:  case  163:  case  164:  case  166:  case  167:  case  170:  case  171:  case  173:  case  174:  case  175:  case  177:  case  178:  case  179:  case  180:  case  181:  case  182:  case  183:  case  184:  case  188:  case  189:  case  190:  case  191:  case  192:  case  193:  case  194:  case  195:  case  196:  case  197:  case  198:  case  199:  case  200:  case  201:  case  204:  case  205:  case  206:  case  207:  case  208:  case  209:  case  210:  case  211:  case  215:  case  216:  case  217:  case  219:  case  220:  case  221:  case  222:  case  223:  case  224:  case  226:  case  227:
    return false;}
    tmp = getLeftModSubIn1Numbers(233,a,b,c,d,e);
    switch(tmp){  case 3:  case  5:  case  6:  case  10:  case  11:  case  12:  case  17:  case  20:  case  21:  case  22:  case  24:  case  27:  case  34:  case  35:  case  39:  case  40:  case  41:  case  42:  case  43:  case  44:  case  45:  case  47:  case  48:  case  53:  case  54:  case  57:  case  59:  case  61:  case  65:  case  67:  case  68:  case  69:  case  70:  case  73:  case  75:  case  77:  case  78:  case  79:  case  80:  case  82:  case  83:  case  84:  case  86:  case  87:  case  88:  case  90:  case  93:  case  94:  case  95:  case  96:  case  97:  case  99:  case  103:  case  106:  case  108:  case  111:  case  114:  case  115:  case  118:  case  119:  case  122:  case  125:  case  127:  case  130:  case  134:  case  136:  case  137:  case  138:  case  139:  case  140:  case  143:  case  145:  case  146:  case  147:  case  149:  case  150:  case  151:  case  153:  case  154:  case  155:  case  156:  case  158:  case  160:  case  163:  case  164:  case  165:  case  166:  case  168:  case  172:  case  174:  case  176:  case  179:  case  180:  case  185:  case  186:  case  188:  case  189:  case  190:  case  191:  case  192:  case  193:  case  194:  case  198:  case  199:  case  206:  case  209:  case  211:  case  212:  case  213:  case  216:  case  221:  case  222:  case  223:  case  227:  case  228:  case  230:
    return false;}
    tmp = getLeftModSubIn1Numbers(239,a,b,c,d,e);
    switch(tmp){  case 7:  case  13:  case  14:  case  19:  case  21:  case  23:  case  26:  case  28:  case  35:  case  37:  case  38:  case  39:  case  41:  case  42:  case  43:  case  46:  case  47:  case  52:  case  53:  case  56:  case  57:  case  59:  case  63:  case  65:  case  69:  case  70:  case  73:  case  74:  case  76:  case  77:  case  78:  case  79:  case  82:  case  84:  case  86:  case  89:  case  92:  case  94:  case  95:  case  97:  case  103:  case  104:  case  105:  case  106:  case  107:  case  111:  case  112:  case  114:  case  115:  case  117:  case  118:  case  119:  case  123:  case  126:  case  129:  case  130:  case  131:  case  137:  case  138:  case  139:  case  140:  case  141:  case  143:  case  146:  case  148:  case  149:  case  151:  case  152:  case  154:  case  156:  case  158:  case  159:  case  164:  case  167:  case  168:  case  171:  case  172:  case  173:  case  175:  case  177:  case  178:  case  179:  case  181:  case  184:  case  185:  case  188:  case  189:  case  190:  case  191:  case  194:  case  195:  case  199:  case  203:  case  205:  case  206:  case  207:  case  208:  case  209:  case  210:  case  212:  case  214:  case  215:  case  217:  case  219:  case  221:  case  222:  case  223:  case  224:  case  227:  case  228:  case  229:  case  230:  case  231:  case  233:  case  234:  case  235:  case  236:  case  237:  case  238:
    return false;}
    tmp = getLeftModSubIn1Numbers(241,a,b,c,d,e);
    switch(tmp){  case 2:  case  3:  case  4:  case  7:  case  9:  case  10:  case  11:  case  12:  case  13:  case  14:  case  15:  case  16:  case  17:  case  18:  case  19:  case  20:  case  21:  case  22:  case  23:  case  24:  case  26:  case  28:  case  29:  case  31:  case  32:  case  33:  case  34:  case  35:  case  37:  case  38:  case  39:  case  42:  case  43:  case  44:  case  45:  case  46:  case  49:  case  50:  case  51:  case  52:  case  53:  case  54:  case  55:  case  56:  case  57:  case  58:  case  59:  case  60:  case  62:  case  63:  case  65:  case  66:  case  67:  case  68:  case  69:  case  70:  case  71:  case  72:  case  73:  case  74:  case  75:  case  76:  case  77:  case  78:  case  80:  case  81:  case  82:  case  83:  case  84:  case  85:  case  86:  case  88:  case  89:  case  90:  case  92:  case  93:  case  94:  case  95:  case  96:  case  97:  case  99:  case  100:  case  101:  case  102:  case  103:  case  104:  case  105:  case  107:  case  108:  case  109:  case  110:  case  111:  case  112:  case  113:  case  114:  case  115:  case  117:  case  118:  case  119:  case  120:  case  121:  case  122:  case  123:  case  124:  case  126:  case  127:  case  128:  case  129:  case  130:  case  131:  case  132:  case  133:  case  134:  case  136:  case  137:  case  138:  case  139:  case  140:  case  141:  case  142:  case  144:  case  145:  case  146:  case  147:  case  148:  case  149:  case  151:  case  152:  case  153:  case  155:  case  156:  case  157:  case  158:  case  159:  case  160:  case  161:  case  163:  case  164:  case  165:  case  166:  case  167:  case  168:  case  169:  case  170:  case  171:  case  172:  case  173:  case  174:  case  175:  case  176:  case  178:  case  179:  case  181:  case  182:  case  183:  case  184:  case  185:  case  186:  case  187:  case  188:  case  189:  case  190:  case  191:  case  192:  case  195:  case  196:  case  197:  case  198:  case  199:  case  202:  case  203:  case  204:  case  206:  case  207:  case  208:  case  209:  case  210:  case  212:  case  213:  case  215:  case  217:  case  218:  case  219:  case  220:  case  221:  case  222:  case  223:  case  224:  case  225:  case  226:  case  227:  case  228:  case  229:  case  230:  case  231:  case  232:  case  234:  case  237:  case  238:  case  239:
    return false;}
    tmp = getLeftModSubIn1Numbers(251,a,b,c,d,e);
    switch(tmp){  case 2:  case  6:  case  8:  case  10:  case  11:  case  14:  case  18:  case  19:  case  24:  case  26:  case  29:  case  30:  case  32:  case  33:  case  34:  case  37:  case  40:  case  42:  case  43:  case  44:  case  46:  case  47:  case  50:  case  53:  case  54:  case  55:  case  56:  case  57:  case  59:  case  61:  case  62:  case  70:  case  71:  case  72:  case  76:  case  77:  case  78:  case  82:  case  87:  case  90:  case  95:  case  96:  case  97:  case  98:  case  99:  case  102:  case  104:  case  107:  case  109:  case  111:  case  116:  case  120:  case  126:  case  127:  case  128:  case  129:  case  130:  case  132:  case  133:  case  134:  case  136:  case  137:  case  138:  case  139:  case  141:  case  143:  case  145:  case  146:  case  148:  case  150:  case  151:  case  157:  case  158:  case  159:  case  160:  case  162:  case  163:  case  165:  case  166:  case  167:  case  168:  case  170:  case  171:  case  172:  case  176:  case  177:  case  178:  case  182:  case  183:  case  184:  case  185:  case  186:  case  187:  case  188:  case  191:  case  193:  case  199:  case  200:  case  202:  case  203:  case  206:  case  210:  case  212:  case  213:  case  215:  case  216:  case  220:  case  223:  case  224:  case  226:  case  228:  case  229:  case  230:  case  231:  case  234:  case  235:  case  236:  case  238:  case  239:  case  242:  case  244:  case  246:  case  247:  case  248:  case  250:
    return false;}
    tmp = getLeftModSubIn1Numbers(257,a,b,c,d,e);
    switch(tmp){  case 3:  case  5:  case  6:  case  7:  case  10:  case  12:  case  14:  case  19:  case  20:  case  24:  case  27:  case  28:  case  33:  case  37:  case  38:  case  39:  case  40:  case  41:  case  43:  case  45:  case  47:  case  48:  case  51:  case  53:  case  54:  case  55:  case  56:  case  63:  case  65:  case  66:  case  69:  case  71:  case  74:  case  75:  case  76:  case  77:  case  78:  case  80:  case  82:  case  83:  case  85:  case  86:  case  87:  case  90:  case  91:  case  93:  case  94:  case  96:  case  97:  case  101:  case  102:  case  103:  case  105:  case  106:  case  107:  case  108:  case  109:  case  110:  case  112:  case  115:  case  119:  case  125:  case  126:  case  127:  case  130:  case  131:  case  132:  case  138:  case  142:  case  145:  case  147:  case  148:  case  149:  case  150:  case  151:  case  152:  case  154:  case  155:  case  156:  case  160:  case  161:  case  163:  case  164:  case  166:  case  167:  case  170:  case  171:  case  172:  case  174:  case  175:  case  177:  case  179:  case  180:  case  181:  case  182:  case  183:  case  186:  case  188:  case  191:  case  192:  case  194:  case  201:  case  202:  case  203:  case  204:  case  206:  case  209:  case  210:  case  212:  case  214:  case  216:  case  217:  case  218:  case  219:  case  220:  case  224:  case  229:  case  230:  case  233:  case  237:  case  238:  case  243:  case  245:  case  247:  case  250:  case  251:  case  252:  case  254:
    return false;}
    tmp = getLeftModSubIn1Numbers(263,a,b,c,d,e);
    switch(tmp){  case 5:  case  7:  case  10:  case  14:  case  15:  case  19:  case  20:  case  21:  case  28:  case  29:  case  30:  case  38:  case  40:  case  41:  case  42:  case  45:  case  47:  case  53:  case  55:  case  56:  case  57:  case  58:  case  59:  case  60:  case  63:  case  65:  case  67:  case  71:  case  73:  case  76:  case  77:  case  79:  case  80:  case  82:  case  84:  case  85:  case  87:  case  90:  case  91:  case  94:  case  97:  case  101:  case  106:  case  107:  case  110:  case  112:  case  113:  case  114:  case  115:  case  116:  case  118:  case  119:  case  120:  case  123:  case  125:  case  126:  case  127:  case  130:  case  131:  case  134:  case  135:  case  139:  case  141:  case  142:  case  146:  case  152:  case  154:  case  155:  case  158:  case  159:  case  160:  case  161:  case  163:  case  164:  case  165:  case  167:  case  168:  case  170:  case  171:  case  174:  case  175:  case  177:  case  180:  case  182:  case  185:  case  188:  case  189:  case  191:  case  193:  case  194:  case  195:  case  197:  case  199:  case  201:  case  202:  case  209:  case  211:  case  212:  case  213:  case  214:  case  215:  case  217:  case  219:  case  220:  case  224:  case  226:  case  227:  case  228:  case  229:  case  230:  case  231:  case  232:  case  236:  case  237:  case  238:  case  239:  case  240:  case  241:  case  245:  case  246:  case  247:  case  250:  case  251:  case  252:  case  254:  case  255:  case  257:  case  259:  case  260:  case  261:  case  262:
    return false;}
    tmp = getLeftModSubIn1Numbers(269,a,b,c,d,e);
    switch(tmp){  case 2:  case  3:  case  7:  case  8:  case  10:  case  12:  case  15:  case  17:  case  18:  case  19:  case  22:  case  26:  case  27:  case  28:  case  29:  case  31:  case  32:  case  33:  case  35:  case  39:  case  40:  case  42:  case  46:  case  48:  case  50:  case  59:  case  60:  case  63:  case  68:  case  69:  case  71:  case  72:  case  74:  case  75:  case  76:  case  77:  case  82:  case  83:  case  85:  case  86:  case  88:  case  90:  case  91:  case  94:  case  95:  case  98:  case  101:  case  102:  case  104:  case  106:  case  107:  case  108:  case  109:  case  110:  case  111:  case  112:  case  113:  case  114:  case  116:  case  122:  case  123:  case  124:  case  128:  case  129:  case  130:  case  132:  case  134:  case  135:  case  137:  case  139:  case  140:  case  141:  case  145:  case  146:  case  147:  case  153:  case  155:  case  156:  case  157:  case  158:  case  159:  case  160:  case  161:  case  162:  case  163:  case  165:  case  167:  case  168:  case  171:  case  174:  case  175:  case  178:  case  179:  case  181:  case  183:  case  184:  case  186:  case  187:  case  192:  case  193:  case  194:  case  195:  case  197:  case  198:  case  200:  case  201:  case  206:  case  209:  case  210:  case  219:  case  221:  case  223:  case  227:  case  229:  case  230:  case  234:  case  236:  case  237:  case  238:  case  240:  case  241:  case  242:  case  243:  case  247:  case  250:  case  251:  case  252:  case  254:  case  257:  case  259:  case  261:  case  262:  case  266:  case  267:
    return false;}
    tmp = getLeftModSubIn1Numbers(271,a,b,c,d,e);
    switch(tmp){  case 2:  case  3:  case  4:  case  5:  case  6:  case  7:  case  11:  case  12:  case  13:  case  14:  case  15:  case  16:  case  17:  case  18:  case  19:  case  20:  case  21:  case  22:  case  23:  case  24:  case  25:  case  26:  case  27:  case  29:  case  30:  case  32:  case  33:  case  36:  case  37:  case  38:  case  40:  case  42:  case  43:  case  45:  case  46:  case  47:  case  48:  case  49:  case  50:  case  51:  case  52:  case  53:  case  54:  case  56:  case  58:  case  59:  case  60:  case  61:  case  62:  case  63:  case  65:  case  66:  case  67:  case  68:  case  70:  case  71:  case  73:  case  74:  case  75:  case  76:  case  77:  case  78:  case  82:  case  83:  case  84:  case  85:  case  86:  case  88:  case  89:  case  91:  case  92:  case  93:  case  94:  case  95:  case  96:  case  97:  case  99:  case  101:  case  102:  case  103:  case  104:  case  105:  case  107:  case  108:  case  109:  case  110:  case  111:  case  112:  case  113:  case  114:  case  115:  case  116:  case  117:  case  118:  case  120:  case  121:  case  122:  case  123:  case  124:  case  126:  case  127:  case  128:  case  129:  case  130:  case  131:  case  132:  case  133:  case  134:  case  135:  case  136:  case  137:  case  138:  case  140:  case  142:  case  143:  case  144:  case  145:  case  146:  case  147:  case  149:  case  150:  case  151:  case  152:  case  153:  case  155:  case  156:  case  157:  case  158:  case  159:  case  160:  case  161:  case  162:  case  163:  case  164:  case  165:  case  168:  case  170:  case  171:  case  172:  case  173:  case  174:  case  175:  case  176:  case  177:  case  179:  case  180:  case  181:  case  182:  case  183:  case  184:  case  186:  case  188:  case  189:  case  190:  case  191:  case  192:  case  193:  case  194:  case  195:  case  196:  case  197:  case  198:  case  199:  case  200:  case  201:  case  202:  case  203:  case  204:  case  205:  case  206:  case  207:  case  208:  case  209:  case  210:  case  211:  case  212:  case  213:  case  214:  case  215:  case  216:  case  217:  case  218:  case  219:  case  220:  case  221:  case  222:  case  223:  case  225:  case  226:  case  227:  case  228:  case  229:  case  230:  case  231:  case  232:  case  233:  case  234:  case  235:  case  236:  case  237:  case  238:  case  239:  case  240:  case  243:  case  245:  case  246:  case  249:  case  250:  case  251:  case  253:  case  254:  case  255:  case  256:  case  257:  case  259:  case  260:  case  261:  case  262:  case  263:  case  264:  case  265:  case  266:  case  267:  case  269:  case  270:
    return false;}
    tmp = getLeftModSubIn1Numbers(277,a,b,c,d,e);
    switch(tmp){  case 2:  case  3:  case  5:  case  6:  case  7:  case  8:  case  9:  case  10:  case  11:  case  12:  case  14:  case  15:  case  17:  case  18:  case  20:  case  22:  case  23:  case  24:  case  25:  case  26:  case  28:  case  29:  case  31:  case  32:  case  33:  case  34:  case  35:  case  36:  case  37:  case  38:  case  39:  case  40:  case  42:  case  43:  case  44:  case  45:  case  46:  case  47:  case  48:  case  49:  case  50:  case  51:  case  53:  case  54:  case  55:  case  56:  case  57:  case  58:  case  60:  case  61:  case  62:  case  63:  case  65:  case  67:  case  68:  case  70:  case  71:  case  72:  case  73:  case  75:  case  77:  case  78:  case  79:  case  80:  case  81:  case  82:  case  83:  case  85:  case  86:  case  87:  case  88:  case  89:  case  90:  case  91:  case  92:  case  93:  case  94:  case  95:  case  96:  case  97:  case  98:  case  99:  case  100:  case  101:  case  103:  case  104:  case  105:  case  106:  case  107:  case  109:  case  110:  case  111:  case  112:  case  114:  case  115:  case  116:  case  117:  case  118:  case  119:  case  121:  case  123:  case  124:  case  125:  case  126:  case  127:  case  128:  case  129:  case  130:  case  132:  case  133:  case  134:  case  135:  case  136:  case  137:  case  138:  case  139:  case  140:  case  141:  case  142:  case  143:  case  144:  case  145:  case  147:  case  148:  case  149:  case  150:  case  151:  case  152:  case  153:  case  154:  case  156:  case  158:  case  159:  case  160:  case  161:  case  162:  case  163:  case  165:  case  166:  case  167:  case  168:  case  170:  case  171:  case  172:  case  173:  case  174:  case  176:  case  177:  case  178:  case  179:  case  180:  case  181:  case  182:  case  183:  case  184:  case  185:  case  186:  case  187:  case  188:  case  189:  case  190:  case  191:  case  192:  case  194:  case  195:  case  196:  case  197:  case  198:  case  199:  case  200:  case  202:  case  204:  case  205:  case  206:  case  207:  case  209:  case  210:  case  212:  case  214:  case  215:  case  216:  case  217:  case  219:  case  220:  case  221:  case  222:  case  223:  case  224:  case  226:  case  227:  case  228:  case  229:  case  230:  case  231:  case  232:  case  233:  case  234:  case  235:  case  237:  case  238:  case  239:  case  240:  case  241:  case  242:  case  243:  case  244:  case  245:  case  246:  case  248:  case  249:  case  251:  case  252:  case  253:  case  254:  case  255:  case  257:  case  259:  case  260:  case  262:  case  263:  case  265:  case  266:  case  267:  case  268:  case  269:  case  270:  case  271:  case  272:  case  274:  case  275:
    return false;}
    tmp = getLeftModSubIn1Numbers(281,a,b,c,d,e);
    switch(tmp){  case 3:  case  6:  case  11:  case  12:  case  13:  case  15:  case  19:  case  21:  case  22:  case  23:  case  24:  case  26:  case  27:  case  30:  case  37:  case  38:  case  41:  case  42:  case  44:  case  46:  case  47:  case  48:  case  51:  case  52:  case  54:  case  55:  case  60:  case  61:  case  65:  case  67:  case  71:  case  73:  case  74:  case  75:  case  76:  case  77:  case  82:  case  83:  case  84:  case  87:  case  88:  case  89:  case  91:  case  92:  case  93:  case  94:  case  95:  case  96:  case  97:  case  99:  case  102:  case  103:  case  104:  case  105:  case  107:  case  108:  case  110:  case  113:  case  115:  case  117:  case  120:  case  122:  case  127:  case  129:  case  130:  case  131:  case  133:  case  134:  case  135:  case  139:  case  142:  case  146:  case  147:  case  148:  case  150:  case  151:  case  152:  case  154:  case  159:  case  161:  case  164:  case  166:  case  168:  case  171:  case  173:  case  174:  case  176:  case  177:  case  178:  case  179:  case  182:  case  184:  case  185:  case  186:  case  187:  case  188:  case  189:  case  190:  case  192:  case  193:  case  194:  case  197:  case  198:  case  199:  case  204:  case  205:  case  206:  case  207:  case  208:  case  210:  case  214:  case  216:  case  220:  case  221:  case  226:  case  227:  case  229:  case  230:  case  233:  case  234:  case  235:  case  237:  case  239:  case  240:  case  243:  case  244:  case  251:  case  254:  case  255:  case  257:  case  258:  case  259:  case  260:  case  262:  case  266:  case  268:  case  269:  case  270:  case  275:  case  278:
    return false;}
    tmp = getLeftModSubIn1Numbers(283,a,b,c,d,e);
    switch(tmp){  case 2:  case  3:  case  5:  case  6:  case  7:  case  8:  case  9:  case  10:  case  11:  case  12:  case  13:  case  14:  case  17:  case  18:  case  19:  case  20:  case  21:  case  22:  case  23:  case  24:  case  25:  case  26:  case  27:  case  28:  case  30:  case  31:  case  32:  case  33:  case  34:  case  35:  case  36:  case  37:  case  39:  case  40:  case  41:  case  43:  case  44:  case  45:  case  46:  case  47:  case  48:  case  49:  case  50:  case  52:  case  53:  case  55:  case  56:  case  57:  case  58:  case  59:  case  62:  case  63:  case  65:  case  67:  case  68:  case  69:  case  70:  case  72:  case  73:  case  74:  case  75:  case  76:  case  77:  case  79:  case  80:  case  81:  case  82:  case  83:  case  84:  case  85:  case  87:  case  88:  case  89:  case  90:  case  91:  case  92:  case  93:  case  94:  case  95:  case  96:  case  97:  case  98:  case  99:  case  100:  case  101:  case  102:  case  103:  case  104:  case  105:  case  107:  case  108:  case  109:  case  110:  case  112:  case  113:  case  114:  case  115:  case  117:  case  118:  case  119:  case  120:  case  121:  case  122:  case  123:  case  124:  case  125:  case  126:  case  128:  case  129:  case  130:  case  131:  case  132:  case  133:  case  135:  case  136:  case  137:  case  138:  case  139:  case  140:  case  142:  case  143:  case  144:  case  145:  case  146:  case  147:  case  148:  case  149:  case  150:  case  153:  case  154:  case  156:  case  157:  case  159:  case  160:  case  162:  case  164:  case  165:  case  166:  case  167:  case  169:  case  170:  case  171:  case  172:  case  173:  case  174:  case  176:  case  177:  case  178:  case  179:  case  180:  case  182:  case  183:  case  184:  case  185:  case  186:  case  187:  case  188:  case  189:  case  190:  case  191:  case  192:  case  193:  case  194:  case  195:  case  196:  case  197:  case  198:  case  200:  case  201:  case  202:  case  203:  case  205:  case  206:  case  208:  case  209:  case  210:  case  211:  case  212:  case  213:  case  214:  case  215:  case  217:  case  218:  case  219:  case  220:  case  221:  case  222:  case  223:  case  224:  case  226:  case  227:  case  228:  case  229:  case  231:  case  232:  case  233:  case  234:  case  235:  case  236:  case  237:  case  238:  case  239:  case  241:  case  242:  case  243:  case  245:  case  246:  case  247:  case  248:  case  249:  case  252:  case  254:  case  255:  case  257:  case  258:  case  259:  case  260:  case  261:  case  263:  case  265:  case  266:  case  267:  case  268:  case  269:  case  270:  case  271:  case  272:  case  273:  case  274:  case  276:  case  277:  case  278:  case  279:  case  280:  case  282:
    return false;}
    tmp = getLeftModSubIn1Numbers(293,a,b,c,d,e);
    switch(tmp){  case 2:  case  3:  case  5:  case  7:  case  8:  case  11:  case  12:  case  13:  case  18:  case  19:  case  20:  case  23:  case  27:  case  28:  case  29:  case  30:  case  32:  case  34:  case  41:  case  42:  case  44:  case  45:  case  47:  case  48:  case  50:  case  51:  case  52:  case  62:  case  63:  case  66:  case  70:  case  72:  case  74:  case  75:  case  76:  case  78:  case  79:  case  80:  case  85:  case  86:  case  89:  case  92:  case  93:  case  98:  case  99:  case  101:  case  103:  case  105:  case  106:  case  108:  case  110:  case  111:  case  112:  case  113:  case  114:  case  116:  case  117:  case  118:  case  119:  case  120:  case  122:  case  125:  case  127:  case  128:  case  129:  case  130:  case  131:  case  134:  case  136:  case  138:  case  139:  case  142:  case  146:  case  147:  case  151:  case  154:  case  155:  case  157:  case  159:  case  162:  case  163:  case  164:  case  165:  case  166:  case  168:  case  171:  case  173:  case  174:  case  175:  case  176:  case  177:  case  179:  case  180:  case  181:  case  182:  case  183:  case  185:  case  187:  case  188:  case  190:  case  192:  case  194:  case  195:  case  200:  case  201:  case  204:  case  207:  case  208:  case  213:  case  214:  case  215:  case  217:  case  218:  case  219:  case  221:  case  223:  case  227:  case  230:  case  231:  case  241:  case  242:  case  243:  case  245:  case  246:  case  248:  case  249:  case  251:  case  252:  case  259:  case  261:  case  263:  case  264:  case  265:  case  266:  case  270:  case  273:  case  274:  case  275:  case  280:  case  281:  case  282:  case  285:  case  286:  case  288:  case  290:  case  291:
    return false;}
    tmp = getLeftModSubIn1Numbers(307,a,b,c,d,e);
    switch(tmp){  case 2:  case  3:  case  5:  case  7:  case  8:  case  10:  case  11:  case  12:  case  13:  case  14:  case  15:  case  18:  case  20:  case  21:  case  22:  case  23:  case  25:  case  26:  case  27:  case  28:  case  29:  case  30:  case  31:  case  32:  case  33:  case  34:  case  35:  case  37:  case  38:  case  39:  case  40:  case  41:  case  42:  case  43:  case  44:  case  45:  case  46:  case  47:  case  48:  case  49:  case  50:  case  51:  case  52:  case  53:  case  55:  case  56:  case  57:  case  58:  case  59:  case  60:  case  61:  case  62:  case  63:  case  65:  case  66:  case  67:  case  69:  case  71:  case  72:  case  73:  case  74:  case  75:  case  78:  case  80:  case  82:  case  83:  case  84:  case  85:  case  86:  case  87:  case  88:  case  89:  case  90:  case  91:  case  92:  case  93:  case  94:  case  95:  case  98:  case  99:  case  100:  case  104:  case  106:  case  107:  case  108:  case  109:  case  110:  case  111:  case  112:  case  116:  case  117:  case  118:  case  119:  case  120:  case  121:  case  122:  case  123:  case  124:  case  125:  case  126:  case  127:  case  128:  case  129:  case  130:  case  131:  case  132:  case  133:  case  134:  case  135:  case  136:  case  137:  case  138:  case  139:  case  140:  case  141:  case  142:  case  143:  case  146:  case  147:  case  148:  case  150:  case  151:  case  152:  case  154:  case  156:  case  157:  case  158:  case  159:  case  160:  case  161:  case  162:  case  163:  case  164:  case  165:  case  166:  case  168:  case  169:  case  170:  case  172:  case  173:  case  174:  case  175:  case  176:  case  177:  case  178:  case  180:  case  181:  case  183:  case  184:  case  185:  case  186:  case  187:  case  188:  case  189:  case  190:  case  191:  case  192:  case  193:  case  194:  case  195:  case  196:  case  197:  case  198:  case  200:  case  201:  case  202:  case  203:  case  204:  case  205:  case  206:  case  207:  case  208:  case  209:  case  210:  case  211:  case  212:  case  213:  case  214:  case  215:  case  217:  case  218:  case  219:  case  220:  case  221:  case  222:  case  223:  case  224:  case  225:  case  226:  case  227:  case  228:  case  229:  case  230:  case  231:  case  232:  case  233:  case  234:  case  236:  case  237:  case  238:  case  239:  case  240:  case  241:  case  242:  case  243:  case  244:  case  245:  case  246:  case  247:  case  248:  case  249:  case  251:  case  252:  case  253:  case  254:  case  255:  case  257:  case  258:  case  260:  case  261:  case  262:  case  263:  case  264:  case  265:  case  266:  case  267:  case  268:  case  270:  case  271:  case  274:  case  276:  case  277:  case  278:  case  279:  case  281:  case  282:  case  283:  case  284:  case  285:  case  286:  case  287:  case  288:  case  290:  case  291:  case  292:  case  293:  case  294:  case  296:  case  297:  case  298:  case  300:  case  301:  case  302:  case  303:  case  306:
    return false;}
    tmp = getLeftModSubIn1Numbers(311,a,b,c,d,e);
    switch(tmp){  case 11:  case  17:  case  19:  case  22:  case  23:  case  29:  case  31:  case  33:  case  34:  case  37:  case  38:  case  41:  case  43:  case  44:  case  46:  case  51:  case  55:  case  57:  case  58:  case  59:  case  61:  case  62:  case  66:  case  68:  case  69:  case  71:  case  74:  case  76:  case  77:  case  82:  case  85:  case  86:  case  87:  case  88:  case  92:  case  93:  case  95:  case  97:  case  99:  case  101:  case  102:  case  103:  case  110:  case  111:  case  114:  case  115:  case  116:  case  118:  case  119:  case  122:  case  123:  case  124:  case  129:  case  131:  case  132:  case  133:  case  136:  case  138:  case  142:  case  143:  case  145:  case  148:  case  149:  case  151:  case  152:  case  153:  case  154:  case  155:  case  161:  case  164:  case  165:  case  167:  case  170:  case  171:  case  172:  case  174:  case  176:  case  177:  case  181:  case  183:  case  184:  case  185:  case  186:  case  190:  case  191:  case  194:  case  198:  case  199:  case  202:  case  203:  case  204:  case  205:  case  206:  case  207:  case  211:  case  213:  case  215:  case  217:  case  220:  case  221:  case  222:  case  227:  case  228:  case  230:  case  231:  case  232:  case  233:  case  236:  case  238:  case  239:  case  241:  case  244:  case  246:  case  247:  case  248:  case  251:  case  255:  case  257:  case  258:  case  259:  case  261:  case  262:  case  263:  case  264:  case  266:  case  269:  case  271:  case  272:  case  275:  case  276:  case  279:  case  281:  case  283:  case  284:  case  285:  case  286:  case  287:  case  290:  case  291:  case  293:  case  295:  case  296:  case  297:  case  298:  case  299:  case  301:  case  302:  case  303:  case  304:  case  305:  case  306:  case  307:  case  308:  case  309:  case  310:
    return false;}
    tmp = getLeftModSubIn1Numbers(313,a,b,c,d,e);
    switch(tmp){  case 2:  case  3:  case  4:  case  5:  case  7:  case  9:  case  10:  case  11:  case  12:  case  13:  case  14:  case  15:  case  16:  case  17:  case  18:  case  20:  case  21:  case  22:  case  23:  case  24:  case  26:  case  28:  case  29:  case  30:  case  31:  case  32:  case  34:  case  37:  case  38:  case  40:  case  41:  case  42:  case  43:  case  45:  case  46:  case  47:  case  50:  case  51:  case  53:  case  54:  case  55:  case  56:  case  57:  case  59:  case  60:  case  61:  case  62:  case  63:  case  65:  case  66:  case  67:  case  68:  case  69:  case  70:  case  72:  case  73:  case  74:  case  75:  case  76:  case  77:  case  78:  case  80:  case  81:  case  82:  case  83:  case  84:  case  85:  case  86:  case  87:  case  88:  case  89:  case  90:  case  91:  case  92:  case  93:  case  94:  case  95:  case  96:  case  98:  case  99:  case  100:  case  101:  case  102:  case  104:  case  105:  case  106:  case  107:  case  108:  case  109:  case  110:  case  111:  case  112:  case  116:  case  117:  case  118:  case  119:  case  120:  case  121:  case  122:  case  123:  case  124:  case  125:  case  126:  case  127:  case  128:  case  129:  case  130:  case  131:  case  132:  case  133:  case  134:  case  135:  case  136:  case  137:  case  138:  case  139:  case  140:  case  141:  case  142:  case  143:  case  144:  case  145:  case  146:  case  147:  case  148:  case  149:  case  153:  case  154:  case  155:  case  156:  case  157:  case  158:  case  159:  case  160:  case  164:  case  165:  case  166:  case  167:  case  168:  case  169:  case  170:  case  171:  case  172:  case  173:  case  174:  case  175:  case  176:  case  177:  case  178:  case  179:  case  180:  case  181:  case  182:  case  183:  case  184:  case  185:  case  186:  case  187:  case  188:  case  189:  case  190:  case  191:  case  192:  case  193:  case  194:  case  195:  case  196:  case  197:  case  201:  case  202:  case  203:  case  204:  case  205:  case  206:  case  207:  case  208:  case  209:  case  211:  case  212:  case  213:  case  214:  case  215:  case  217:  case  218:  case  219:  case  220:  case  221:  case  222:  case  223:  case  224:  case  225:  case  226:  case  227:  case  228:  case  229:  case  230:  case  231:  case  232:  case  233:  case  235:  case  236:  case  237:  case  238:  case  239:  case  240:  case  241:  case  243:  case  244:  case  245:  case  246:  case  247:  case  248:  case  250:  case  251:  case  252:  case  253:  case  254:  case  256:  case  257:  case  258:  case  259:  case  260:  case  262:  case  263:  case  266:  case  267:  case  268:  case  270:  case  271:  case  272:  case  273:  case  275:  case  276:  case  279:  case  281:  case  282:  case  283:  case  284:  case  285:  case  287:  case  289:  case  290:  case  291:  case  292:  case  293:  case  295:  case  296:  case  297:  case  298:  case  299:  case  300:  case  301:  case  302:  case  303:  case  304:  case  306:  case  308:  case  309:  case  310:  case  311:
    return false;}
    tmp = getLeftModSubIn1Numbers(317,a,b,c,d,e);
    switch(tmp){  case 2:  case  3:  case  5:  case  8:  case  12:  case  13:  case  14:  case  17:  case  18:  case  19:  case  20:  case  21:  case  22:  case  27:  case  29:  case  30:  case  32:  case  33:  case  35:  case  41:  case  45:  case  46:  case  47:  case  48:  case  50:  case  52:  case  55:  case  56:  case  62:  case  68:  case  69:  case  71:  case  72:  case  74:  case  75:  case  76:  case  78:  case  80:  case  84:  case  86:  case  88:  case  91:  case  93:  case  97:  case  98:  case  102:  case  106:  case  107:  case  108:  case  109:  case  111:  case  114:  case  115:  case  116:  case  117:  case  118:  case  119:  case  120:  case  122:  case  125:  case  126:  case  127:  case  128:  case  129:  case  130:  case  132:  case  133:  case  134:  case  137:  case  139:  case  140:  case  143:  case  146:  case  147:  case  151:  case  153:  case  154:  case  155:  case  158:  case  159:  case  162:  case  163:  case  164:  case  166:  case  170:  case  171:  case  174:  case  177:  case  178:  case  180:  case  183:  case  184:  case  185:  case  187:  case  188:  case  189:  case  190:  case  191:  case  192:  case  195:  case  197:  case  198:  case  199:  case  200:  case  201:  case  202:  case  203:  case  206:  case  208:  case  209:  case  210:  case  211:  case  215:  case  219:  case  220:  case  224:  case  226:  case  229:  case  231:  case  233:  case  237:  case  239:  case  241:  case  242:  case  243:  case  245:  case  246:  case  248:  case  249:  case  255:  case  261:  case  262:  case  265:  case  267:  case  269:  case  270:  case  271:  case  272:  case  276:  case  282:  case  284:  case  285:  case  287:  case  288:  case  290:  case  295:  case  296:  case  297:  case  298:  case  299:  case  300:  case  303:  case  304:  case  305:  case  309:  case  312:  case  314:  case  315:
    return false;}
    tmp = getLeftModSubIn1Numbers(331,a,b,c,d,e);
    switch(tmp){  case 2:  case  3:  case  4:  case  5:  case  6:  case  7:  case  8:  case  9:  case  10:  case  11:  case  12:  case  13:  case  14:  case  15:  case  16:  case  17:  case  18:  case  19:  case  20:  case  21:  case  23:  case  24:  case  25:  case  26:  case  27:  case  28:  case  29:  case  30:  case  31:  case  32:  case  34:  case  35:  case  36:  case  37:  case  38:  case  39:  case  40:  case  41:  case  42:  case  43:  case  44:  case  45:  case  46:  case  47:  case  48:  case  50:  case  51:  case  52:  case  53:  case  54:  case  55:  case  57:  case  58:  case  59:  case  60:  case  61:  case  62:  case  63:  case  65:  case  66:  case  69:  case  71:  case  72:  case  73:  case  75:  case  76:  case  77:  case  78:  case  79:  case  81:  case  82:  case  83:  case  86:  case  87:  case  88:  case  89:  case  90:  case  91:  case  92:  case  93:  case  94:  case  95:  case  97:  case  98:  case  99:  case  101:  case  103:  case  104:  case  106:  case  107:  case  108:  case  109:  case  110:  case  112:  case  113:  case  114:  case  115:  case  116:  case  117:  case  119:  case  121:  case  122:  case  123:  case  127:  case  128:  case  129:  case  130:  case  131:  case  132:  case  133:  case  134:  case  135:  case  136:  case  137:  case  138:  case  139:  case  140:  case  141:  case  142:  case  145:  case  146:  case  147:  case  148:  case  149:  case  151:  case  152:  case  154:  case  156:  case  158:  case  159:  case  160:  case  161:  case  162:  case  163:  case  164:  case  165:  case  166:  case  168:  case  169:  case  170:  case  171:  case  173:  case  174:  case  175:  case  176:  case  178:  case  179:  case  181:  case  182:  case  183:  case  184:  case  185:  case  187:  case  188:  case  190:  case  191:  case  192:  case  194:  case  195:  case  196:  case  197:  case  198:  case  199:  case  200:  case  201:  case  202:  case  203:  case  204:  case  205:  case  206:  case  207:  case  208:  case  209:  case  210:  case  211:  case  212:  case  213:  case  217:  case  218:  case  219:  case  220:  case  221:  case  222:  case  223:  case  224:  case  226:  case  227:  case  228:  case  229:  case  230:  case  231:  case  232:  case  233:  case  234:  case  235:  case  236:  case  237:  case  238:  case  240:  case  241:  case  242:  case  243:  case  244:  case  245:  case  246:  case  247:  case  248:  case  249:  case  250:  case  251:  case  252:  case  254:  case  255:  case  256:  case  257:  case  259:  case  260:  case  261:  case  262:  case  263:  case  264:  case  265:  case  267:  case  268:  case  269:  case  271:  case  272:  case  273:  case  275:  case  276:  case  277:  case  278:  case  280:  case  281:  case  282:  case  283:  case  284:  case  285:  case  286:  case  287:  case  288:  case  289:  case  290:  case  291:  case  292:  case  294:  case  295:  case  296:  case  297:  case  298:  case  299:  case  300:  case  301:  case  302:  case  303:  case  305:  case  306:  case  307:  case  308:  case  309:  case  310:  case  311:  case  312:  case  314:  case  315:  case  317:  case  318:  case  320:  case  322:  case  325:  case  326:  case  327:  case  328:  case  329:  case  330:
    return false;}
    tmp = getLeftModSubIn1Numbers(337,a,b,c,d,e);
    switch(tmp){  case 2:  case  3:  case  4:  case  5:  case  9:  case  10:  case  11:  case  12:  case  13:  case  14:  case  15:  case  16:  case  17:  case  18:  case  19:  case  20:  case  21:  case  22:  case  23:  case  24:  case  26:  case  28:  case  29:  case  30:  case  31:  case  32:  case  33:  case  34:  case  35:  case  37:  case  38:  case  40:  case  41:  case  44:  case  45:  case  46:  case  50:  case  51:  case  53:  case  54:  case  57:  case  58:  case  59:  case  60:  case  61:  case  62:  case  63:  case  65:  case  66:  case  67:  case  68:  case  69:  case  70:  case  71:  case  72:  case  73:  case  74:  case  75:  case  76:  case  77:  case  78:  case  80:  case  81:  case  82:  case  83:  case  84:  case  86:  case  87:  case  88:  case  89:  case  90:  case  91:  case  92:  case  93:  case  94:  case  95:  case  96:  case  97:  case  98:  case  99:  case  100:  case  101:  case  102:  case  104:  case  105:  case  106:  case  107:  case  108:  case  109:  case  110:  case  112:  case  113:  case  114:  case  115:  case  116:  case  117:  case  118:  case  119:  case  120:  case  122:  case  124:  case  125:  case  126:  case  127:  case  128:  case  129:  case  130:  case  131:  case  132:  case  133:  case  134:  case  135:  case  136:  case  138:  case  139:  case  140:  case  141:  case  142:  case  143:  case  144:  case  145:  case  146:  case  147:  case  149:  case  151:  case  152:  case  153:  case  154:  case  155:  case  156:  case  157:  case  158:  case  159:  case  160:  case  161:  case  163:  case  165:  case  166:  case  167:  case  168:  case  169:  case  170:  case  171:  case  172:  case  174:  case  176:  case  177:  case  178:  case  179:  case  180:  case  181:  case  182:  case  183:  case  184:  case  185:  case  186:  case  188:  case  190:  case  191:  case  192:  case  193:  case  194:  case  195:  case  196:  case  197:  case  198:  case  199:  case  201:  case  202:  case  203:  case  204:  case  205:  case  206:  case  207:  case  208:  case  209:  case  210:  case  211:  case  212:  case  213:  case  215:  case  217:  case  218:  case  219:  case  220:  case  221:  case  222:  case  223:  case  224:  case  225:  case  227:  case  228:  case  229:  case  230:  case  231:  case  232:  case  233:  case  235:  case  236:  case  237:  case  238:  case  239:  case  240:  case  241:  case  242:  case  243:  case  244:  case  245:  case  246:  case  247:  case  248:  case  249:  case  250:  case  251:  case  253:  case  254:  case  255:  case  256:  case  257:  case  259:  case  260:  case  261:  case  262:  case  263:  case  264:  case  265:  case  266:  case  267:  case  268:  case  269:  case  270:  case  271:  case  272:  case  274:  case  275:  case  276:  case  277:  case  278:  case  279:  case  280:  case  283:  case  284:  case  286:  case  287:  case  291:  case  292:  case  293:  case  296:  case  297:  case  299:  case  300:  case  302:  case  303:  case  304:  case  305:  case  306:  case  307:  case  308:  case  309:  case  311:  case  313:  case  314:  case  315:  case  316:  case  317:  case  318:  case  319:  case  320:  case  321:  case  322:  case  323:  case  324:  case  325:  case  326:  case  327:  case  328:  case  332:  case  333:  case  334:  case  335:
    return false;}
    tmp = getLeftModSubIn1Numbers(347,a,b,c,d,e);
    switch(tmp){  case 2:  case  5:  case  6:  case  7:  case  8:  case  15:  case  17:  case  18:  case  19:  case  20:  case  21:  case  22:  case  23:  case  24:  case  26:  case  28:  case  32:  case  37:  case  41:  case  45:  case  47:  case  50:  case  51:  case  54:  case  55:  case  57:  case  58:  case  60:  case  62:  case  63:  case  65:  case  66:  case  68:  case  69:  case  70:  case  72:  case  76:  case  77:  case  78:  case  79:  case  80:  case  84:  case  86:  case  88:  case  91:  case  92:  case  96:  case  97:  case  98:  case  101:  case  103:  case  104:  case  106:  case  111:  case  112:  case  118:  case  122:  case  123:  case  125:  case  128:  case  134:  case  135:  case  139:  case  141:  case  142:  case  145:  case  146:  case  148:  case  150:  case  151:  case  153:  case  155:  case  162:  case  163:  case  164:  case  165:  case  166:  case  170:  case  171:  case  174:  case  175:  case  178:  case  179:  case  180:  case  186:  case  187:  case  188:  case  189:  case  190:  case  191:  case  193:  case  195:  case  198:  case  200:  case  203:  case  204:  case  207:  case  209:  case  210:  case  211:  case  214:  case  215:  case  216:  case  217:  case  218:  case  220:  case  221:  case  223:  case  226:  case  227:  case  228:  case  230:  case  231:  case  232:  case  233:  case  234:  case  237:  case  238:  case  239:  case  240:  case  242:  case  245:  case  247:  case  248:  case  252:  case  253:  case  254:  case  257:  case  258:  case  260:  case  262:  case  264:  case  265:  case  266:  case  272:  case  273:  case  274:  case  276:  case  280:  case  283:  case  286:  case  288:  case  291:  case  294:  case  295:  case  298:  case  299:  case  301:  case  303:  case  304:  case  305:  case  307:  case  308:  case  309:  case  311:  case  312:  case  313:  case  314:  case  316:  case  317:  case  318:  case  320:  case  322:  case  331:  case  333:  case  334:  case  335:  case  336:  case  337:  case  338:  case  343:  case  344:  case  346:
    return false;}
    tmp = getLeftModSubIn1Numbers(349,a,b,c,d,e);
    switch(tmp){  case 2:  case  3:  case  4:  case  5:  case  6:  case  7:  case  8:  case  9:  case  10:  case  11:  case  12:  case  13:  case  14:  case  15:  case  16:  case  18:  case  19:  case  20:  case  21:  case  22:  case  23:  case  24:  case  25:  case  26:  case  28:  case  29:  case  30:  case  32:  case  33:  case  34:  case  35:  case  38:  case  39:  case  40:  case  42:  case  43:  case  44:  case  46:  case  47:  case  49:  case  50:  case  51:  case  52:  case  53:  case  54:  case  55:  case  56:  case  57:  case  58:  case  59:  case  61:  case  62:  case  63:  case  65:  case  68:  case  70:  case  71:  case  72:  case  73:  case  74:  case  76:  case  77:  case  78:  case  79:  case  81:  case  82:  case  83:  case  84:  case  85:  case  87:  case  89:  case  90:  case  91:  case  93:  case  94:  case  95:  case  96:  case  97:  case  98:  case  99:  case  101:  case  102:  case  103:  case  104:  case  105:  case  106:  case  107:  case  108:  case  109:  case  111:  case  112:  case  113:  case  114:  case  116:  case  117:  case  119:  case  120:  case  122:  case  123:  case  124:  case  127:  case  128:  case  129:  case  130:  case  131:  case  132:  case  133:  case  134:  case  135:  case  136:  case  137:  case  138:  case  140:  case  141:  case  142:  case  143:  case  144:  case  145:  case  146:  case  147:  case  148:  case  149:  case  150:  case  151:  case  152:  case  153:  case  154:  case  155:  case  156:  case  157:  case  158:  case  159:  case  160:  case  161:  case  162:  case  163:  case  164:  case  165:  case  166:  case  167:  case  169:  case  170:  case  172:  case  173:  case  174:  case  175:  case  176:  case  177:  case  179:  case  180:  case  182:  case  183:  case  184:  case  185:  case  186:  case  187:  case  188:  case  189:  case  190:  case  191:  case  192:  case  193:  case  194:  case  195:  case  196:  case  197:  case  198:  case  199:  case  200:  case  201:  case  202:  case  203:  case  204:  case  205:  case  206:  case  207:  case  208:  case  209:  case  211:  case  212:  case  213:  case  214:  case  215:  case  216:  case  217:  case  218:  case  219:  case  220:  case  221:  case  222:  case  225:  case  226:  case  227:  case  229:  case  230:  case  232:  case  233:  case  235:  case  236:  case  237:  case  238:  case  240:  case  241:  case  242:  case  243:  case  244:  case  245:  case  246:  case  247:  case  248:  case  250:  case  251:  case  252:  case  253:  case  254:  case  255:  case  256:  case  258:  case  259:  case  260:  case  262:  case  264:  case  265:  case  266:  case  267:  case  268:  case  270:  case  271:  case  272:  case  273:  case  275:  case  276:  case  277:  case  278:  case  279:  case  281:  case  284:  case  286:  case  287:  case  288:  case  290:  case  291:  case  292:  case  293:  case  294:  case  295:  case  296:  case  297:  case  298:  case  299:  case  300:  case  302:  case  303:  case  305:  case  306:  case  307:  case  309:  case  310:  case  311:  case  314:  case  315:  case  316:  case  317:  case  319:  case  320:  case  321:  case  323:  case  324:  case  325:  case  326:  case  327:  case  328:  case  329:  case  330:  case  331:  case  333:  case  334:  case  335:  case  336:  case  337:  case  338:  case  339:  case  340:  case  341:  case  342:  case  343:  case  344:  case  345:  case  346:  case  347:
    return false;}
    tmp = getLeftModSubIn1Numbers(353,a,b,c,d,e);
    switch(tmp){  case 3:  case  5:  case  6:  case  7:  case  10:  case  12:  case  13:  case  14:  case  20:  case  24:  case  26:  case  27:  case  28:  case  31:  case  33:  case  37:  case  40:  case  45:  case  48:  case  51:  case  52:  case  53:  case  54:  case  55:  case  56:  case  57:  case  59:  case  62:  case  63:  case  66:  case  67:  case  69:  case  71:  case  74:  case  75:  case  77:  case  79:  case  80:  case  85:  case  87:  case  89:  case  90:  case  95:  case  96:  case  101:  case  102:  case  103:  case  104:  case  105:  case  106:  case  107:  case  108:  case  110:  case  112:  case  114:  case  115:  case  117:  case  118:  case  119:  case  123:  case  124:  case  125:  case  126:  case  129:  case  132:  case  133:  case  134:  case  137:  case  138:  case  139:  case  141:  case  142:  case  143:  case  145:  case  147:  case  148:  case  149:  case  150:  case  151:  case  154:  case  158:  case  160:  case  161:  case  163:  case  170:  case  173:  case  174:  case  175:  case  178:  case  179:  case  180:  case  183:  case  190:  case  192:  case  193:  case  195:  case  199:  case  202:  case  203:  case  204:  case  205:  case  206:  case  208:  case  210:  case  211:  case  212:  case  214:  case  215:  case  216:  case  219:  case  220:  case  221:  case  224:  case  227:  case  228:  case  229:  case  230:  case  234:  case  235:  case  236:  case  238:  case  239:  case  241:  case  243:  case  245:  case  246:  case  247:  case  248:  case  249:  case  250:  case  251:  case  252:  case  257:  case  258:  case  263:  case  264:  case  266:  case  268:  case  273:  case  274:  case  276:  case  278:  case  279:  case  282:  case  284:  case  286:  case  287:  case  290:  case  291:  case  294:  case  296:  case  297:  case  298:  case  299:  case  300:  case  301:  case  302:  case  305:  case  308:  case  313:  case  316:  case  320:  case  322:  case  325:  case  326:  case  327:  case  329:  case  333:  case  339:  case  340:  case  341:  case  343:  case  346:  case  347:  case  348:  case  350:
    return false;}
    tmp = getLeftModSubIn1Numbers(359,a,b,c,d,e);
    switch(tmp){  case 7:  case  13:  case  14:  case  19:  case  21:  case  26:  case  28:  case  29:  case  31:  case  35:  case  38:  case  39:  case  42:  case  43:  case  52:  case  53:  case  56:  case  57:  case  58:  case  59:  case  61:  case  62:  case  63:  case  65:  case  67:  case  70:  case  71:  case  76:  case  77:  case  78:  case  83:  case  84:  case  86:  case  87:  case  89:  case  93:  case  95:  case  97:  case  103:  case  104:  case  105:  case  106:  case  109:  case  112:  case  113:  case  114:  case  116:  case  117:  case  118:  case  119:  case  122:  case  124:  case  126:  case  129:  case  130:  case  134:  case  137:  case  139:  case  140:  case  142:  case  143:  case  145:  case  152:  case  154:  case  155:  case  156:  case  157:  case  159:  case  161:  case  163:  case  166:  case  167:  case  168:  case  171:  case  172:  case  174:  case  175:  case  177:  case  178:  case  179:  case  183:  case  186:  case  189:  case  190:  case  194:  case  195:  case  197:  case  199:  case  201:  case  206:  case  208:  case  209:  case  210:  case  211:  case  212:  case  213:  case  215:  case  218:  case  221:  case  223:  case  224:  case  226:  case  227:  case  228:  case  231:  case  232:  case  234:  case  236:  case  238:  case  239:  case  244:  case  248:  case  249:  case  251:  case  252:  case  257:  case  258:  case  259:  case  260:  case  261:  case  263:  case  265:  case  267:  case  268:  case  269:  case  271:  case  274:  case  277:  case  278:  case  279:  case  280:  case  284:  case  285:  case  286:  case  287:  case  290:  case  291:  case  293:  case  295:  case  299:  case  304:  case  305:  case  308:  case  309:  case  310:  case  311:  case  312:  case  313:  case  314:  case  315:  case  318:  case  319:  case  322:  case  323:  case  325:  case  326:  case  327:  case  329:  case  332:  case  334:  case  335:  case  336:  case  337:  case  339:  case  341:  case  342:  case  343:  case  344:  case  347:  case  348:  case  349:  case  350:  case  351:  case  353:  case  354:  case  355:  case  356:  case  357:  case  358:
    return false;}
    tmp = getLeftModSubIn1Numbers(367,a,b,c,d,e);
    switch(tmp){  case 2:  case  3:  case  4:  case  5:  case  6:  case  10:  case  11:  case  12:  case  13:  case  14:  case  16:  case  17:  case  18:  case  19:  case  20:  case  21:  case  22:  case  23:  case  24:  case  26:  case  27:  case  28:  case  29:  case  30:  case  31:  case  32:  case  33:  case  34:  case  35:  case  36:  case  37:  case  38:  case  39:  case  40:  case  41:  case  42:  case  43:  case  44:  case  45:  case  48:  case  50:  case  51:  case  53:  case  54:  case  55:  case  57:  case  58:  case  60:  case  61:  case  62:  case  65:  case  66:  case  67:  case  68:  case  69:  case  70:  case  71:  case  73:  case  75:  case  76:  case  77:  case  78:  case  79:  case  80:  case  82:  case  83:  case  84:  case  85:  case  86:  case  88:  case  89:  case  90:  case  91:  case  92:  case  93:  case  94:  case  95:  case  96:  case  97:  case  98:  case  99:  case  100:  case  102:  case  103:  case  104:  case  108:  case  109:  case  110:  case  111:  case  112:  case  113:  case  115:  case  116:  case  117:  case  118:  case  119:  case  121:  case  123:  case  125:  case  126:  case  127:  case  128:  case  129:  case  130:  case  131:  case  133:  case  136:  case  138:  case  139:  case  140:  case  141:  case  142:  case  143:  case  144:  case  146:  case  147:  case  148:  case  149:  case  150:  case  152:  case  153:  case  154:  case  155:  case  156:  case  157:  case  158:  case  159:  case  160:  case  161:  case  162:  case  163:  case  165:  case  166:  case  167:  case  168:  case  169:  case  170:  case  171:  case  172:  case  173:  case  174:  case  176:  case  177:  case  179:  case  180:  case  181:  case  182:  case  183:  case  184:  case  185:  case  186:  case  187:  case  188:  case  189:  case  191:  case  192:  case  193:  case  194:  case  195:  case  196:  case  197:  case  198:  case  201:  case  202:  case  203:  case  205:  case  206:  case  207:  case  208:  case  210:  case  212:  case  213:  case  214:  case  215:  case  216:  case  217:  case  218:  case  219:  case  221:  case  222:  case  223:  case  224:  case  227:  case  228:  case  230:  case  231:  case  232:  case  233:  case  234:  case  235:  case  236:  case  237:  case  238:  case  239:  case  240:  case  241:  case  243:  case  244:  case  245:  case  246:  case  247:  case  248:  case  249:  case  250:  case  251:  case  252:  case  253:  case  254:  case  255:  case  256:  case  257:  case  259:  case  260:  case  261:  case  262:  case  263:  case  264:  case  265:  case  266:  case  267:  case  268:  case  269:  case  270:  case  271:  case  272:  case  273:  case  274:  case  275:  case  276:  case  277:  case  278:  case  279:  case  280:  case  282:  case  283:  case  284:  case  285:  case  286:  case  287:  case  288:  case  289:  case  290:  case  291:  case  293:  case  294:  case  295:  case  296:  case  297:  case  298:  case  300:  case  301:  case  302:  case  303:  case  304:  case  305:  case  306:  case  307:  case  308:  case  309:  case  310:  case  311:  case  312:  case  313:  case  314:  case  315:  case  316:  case  317:  case  318:  case  319:  case  320:  case  321:  case  324:  case  325:  case  326:  case  328:  case  330:  case  331:  case  333:  case  334:  case  335:  case  336:  case  337:  case  339:  case  341:  case  342:  case  344:  case  345:  case  347:  case  348:  case  349:  case  350:  case  351:  case  352:  case  353:  case  354:  case  355:  case  356:  case  357:  case  358:  case  359:  case  360:  case  361:  case  363:  case  365:  case  366:
    return false;}
    tmp = getLeftModSubIn1Numbers(373,a,b,c,d,e);
    switch(tmp){  case 2:  case  3:  case  4:  case  5:  case  6:  case  8:  case  9:  case  10:  case  11:  case  14:  case  15:  case  16:  case  18:  case  19:  case  20:  case  21:  case  23:  case  24:  case  25:  case  26:  case  28:  case  29:  case  32:  case  33:  case  34:  case  35:  case  36:  case  37:  case  38:  case  39:  case  40:  case  42:  case  43:  case  44:  case  45:  case  46:  case  47:  case  48:  case  50:  case  51:  case  52:  case  53:  case  54:  case  56:  case  57:  case  58:  case  59:  case  60:  case  61:  case  62:  case  63:  case  65:  case  66:  case  67:  case  68:  case  69:  case  70:  case  71:  case  72:  case  73:  case  74:  case  76:  case  77:  case  78:  case  79:  case  80:  case  81:  case  82:  case  83:  case  85:  case  88:  case  89:  case  90:  case  92:  case  93:  case  94:  case  95:  case  96:  case  97:  case  98:  case  99:  case  100:  case  101:  case  102:  case  103:  case  104:  case  105:  case  106:  case  107:  case  108:  case  110:  case  112:  case  113:  case  114:  case  115:  case  116:  case  117:  case  118:  case  120:  case  121:  case  122:  case  123:  case  124:  case  125:  case  126:  case  127:  case  128:  case  129:  case  130:  case  131:  case  132:  case  133:  case  134:  case  135:  case  136:  case  138:  case  139:  case  140:  case  141:  case  142:  case  143:  case  145:  case  146:  case  147:  case  148:  case  149:  case  150:  case  151:  case  153:  case  155:  case  157:  case  159:  case  161:  case  162:  case  164:  case  165:  case  166:  case  167:  case  168:  case  170:  case  171:  case  172:  case  173:  case  174:  case  175:  case  176:  case  177:  case  178:  case  179:  case  180:  case  181:  case  182:  case  183:  case  185:  case  186:  case  187:  case  188:  case  190:  case  191:  case  192:  case  193:  case  194:  case  195:  case  196:  case  197:  case  198:  case  199:  case  200:  case  201:  case  202:  case  203:  case  205:  case  206:  case  207:  case  208:  case  209:  case  211:  case  212:  case  214:  case  216:  case  218:  case  220:  case  222:  case  223:  case  224:  case  225:  case  226:  case  227:  case  228:  case  230:  case  231:  case  232:  case  233:  case  234:  case  235:  case  237:  case  238:  case  239:  case  240:  case  241:  case  242:  case  243:  case  244:  case  245:  case  246:  case  247:  case  248:  case  249:  case  250:  case  251:  case  252:  case  253:  case  255:  case  256:  case  257:  case  258:  case  259:  case  260:  case  261:  case  263:  case  265:  case  266:  case  267:  case  268:  case  269:  case  270:  case  271:  case  272:  case  273:  case  274:  case  275:  case  276:  case  277:  case  278:  case  279:  case  280:  case  281:  case  283:  case  284:  case  285:  case  288:  case  290:  case  291:  case  292:  case  293:  case  294:  case  295:  case  296:  case  297:  case  299:  case  300:  case  301:  case  302:  case  303:  case  304:  case  305:  case  306:  case  307:  case  308:  case  310:  case  311:  case  312:  case  313:  case  314:  case  315:  case  316:  case  317:  case  319:  case  320:  case  321:  case  322:  case  323:  case  325:  case  326:  case  327:  case  328:  case  329:  case  330:  case  331:  case  333:  case  334:  case  335:  case  336:  case  337:  case  338:  case  339:  case  340:  case  341:  case  344:  case  345:  case  347:  case  348:  case  349:  case  350:  case  352:  case  353:  case  354:  case  355:  case  357:  case  358:  case  359:  case  362:  case  363:  case  364:  case  365:  case  367:  case  368:  case  369:  case  370:  case  371:
    return false;}
    tmp = getLeftModSubIn1Numbers(379,a,b,c,d,e);
    switch(tmp){  case 2:  case  3:  case  4:  case  7:  case  8:  case  9:  case  10:  case  11:  case  12:  case  13:  case  15:  case  16:  case  17:  case  18:  case  19:  case  20:  case  21:  case  22:  case  24:  case  26:  case  27:  case  28:  case  29:  case  31:  case  32:  case  34:  case  35:  case  38:  case  40:  case  42:  case  43:  case  44:  case  45:  case  46:  case  47:  case  48:  case  49:  case  50:  case  52:  case  53:  case  54:  case  55:  case  56:  case  57:  case  58:  case  59:  case  60:  case  61:  case  62:  case  63:  case  65:  case  66:  case  68:  case  69:  case  71:  case  72:  case  73:  case  74:  case  75:  case  78:  case  79:  case  80:  case  81:  case  82:  case  85:  case  87:  case  88:  case  89:  case  90:  case  92:  case  95:  case  96:  case  97:  case  98:  case  99:  case  100:  case  101:  case  102:  case  103:  case  104:  case  105:  case  106:  case  107:  case  108:  case  109:  case  110:  case  111:  case  112:  case  113:  case  114:  case  116:  case  117:  case  118:  case  120:  case  121:  case  122:  case  123:  case  124:  case  126:  case  127:  case  128:  case  129:  case  130:  case  131:  case  132:  case  133:  case  134:  case  135:  case  136:  case  140:  case  141:  case  143:  case  144:  case  145:  case  146:  case  147:  case  148:  case  149:  case  151:  case  152:  case  153:  case  154:  case  155:  case  156:  case  157:  case  158:  case  160:  case  161:  case  162:  case  163:  case  164:  case  166:  case  168:  case  169:  case  170:  case  171:  case  172:  case  173:  case  174:  case  175:  case  176:  case  177:  case  178:  case  181:  case  182:  case  183:  case  184:  case  186:  case  187:  case  188:  case  189:  case  190:  case  191:  case  192:  case  193:  case  194:  case  197:  case  199:  case  200:  case  201:  case  202:  case  203:  case  204:  case  206:  case  207:  case  208:  case  209:  case  210:  case  211:  case  212:  case  213:  case  214:  case  215:  case  218:  case  219:  case  220:  case  221:  case  223:  case  224:  case  225:  case  226:  case  227:  case  228:  case  229:  case  230:  case  231:  case  233:  case  235:  case  236:  case  237:  case  238:  case  239:  case  240:  case  241:  case  242:  case  243:  case  245:  case  247:  case  248:  case  249:  case  250:  case  251:  case  252:  case  253:  case  254:  case  256:  case  257:  case  258:  case  259:  case  260:  case  261:  case  262:  case  263:  case  264:  case  265:  case  266:  case  268:  case  269:  case  270:  case  271:  case  272:  case  273:  case  274:  case  275:  case  276:  case  277:  case  278:  case  279:  case  280:  case  281:  case  282:  case  283:  case  284:  case  285:  case  286:  case  287:  case  288:  case  289:  case  290:  case  291:  case  292:  case  293:  case  294:  case  295:  case  296:  case  297:  case  298:  case  299:  case  300:  case  301:  case  302:  case  303:  case  304:  case  305:  case  307:  case  308:  case  309:  case  310:  case  312:  case  313:  case  314:  case  315:  case  317:  case  318:  case  319:  case  321:  case  323:  case  324:  case  325:  case  326:  case  328:  case  329:  case  330:  case  332:  case  333:  case  334:  case  336:  case  337:  case  338:  case  340:  case  341:  case  342:  case  343:  case  344:  case  345:  case  346:  case  347:  case  348:  case  349:  case  351:  case  353:  case  354:  case  355:  case  356:  case  357:  case  358:  case  359:  case  360:  case  361:  case  362:  case  363:  case  364:  case  365:  case  366:  case  367:  case  368:  case  369:  case  370:  case  372:  case  373:  case  374:  case  375:  case  376:  case  377:  case  378:
    return false;}
    tmp = getLeftModSubIn1Numbers(383,a,b,c,d,e);
    switch(tmp){  case 5:  case  10:  case  11:  case  13:  case  15:  case  20:  case  22:  case  26:  case  30:  case  33:  case  35:  case  37:  case  39:  case  40:  case  41:  case  44:  case  45:  case  47:  case  52:  case  53:  case  59:  case  60:  case  61:  case  66:  case  70:  case  74:  case  77:  case  78:  case  79:  case  80:  case  82:  case  83:  case  85:  case  88:  case  89:  case  90:  case  91:  case  94:  case  95:  case  97:  case  99:  case  104:  case  105:  case  106:  case  107:  case  109:  case  111:  case  115:  case  117:  case  118:  case  120:  case  122:  case  123:  case  125:  case  127:  case  131:  case  132:  case  135:  case  140:  case  141:  case  145:  case  148:  case  151:  case  154:  case  155:  case  156:  case  157:  case  158:  case  159:  case  160:  case  163:  case  164:  case  166:  case  167:  case  170:  case  176:  case  177:  case  178:  case  179:  case  180:  case  181:  case  182:  case  183:  case  187:  case  188:  case  190:  case  191:  case  194:  case  197:  case  198:  case  199:  case  208:  case  209:  case  210:  case  211:  case  212:  case  214:  case  215:  case  218:  case  221:  case  222:  case  230:  case  231:  case  233:  case  234:  case  236:  case  237:  case  239:  case  240:  case  241:  case  244:  case  245:  case  246:  case  247:  case  249:  case  250:  case  253:  case  254:  case  255:  case  257:  case  259:  case  262:  case  264:  case  267:  case  269:  case  270:  case  271:  case  273:  case  275:  case  280:  case  281:  case  282:  case  283:  case  285:  case  287:  case  290:  case  291:  case  296:  case  297:  case  299:  case  302:  case  307:  case  308:  case  310:  case  311:  case  312:  case  314:  case  315:  case  316:  case  318:  case  319:  case  320:  case  321:  case  325:  case  326:  case  327:  case  328:  case  329:  case  332:  case  333:  case  334:  case  335:  case  337:  case  340:  case  341:  case  345:  case  347:  case  349:  case  351:  case  352:  case  354:  case  355:  case  356:  case  358:  case  359:  case  360:  case  362:  case  364:  case  365:  case  366:  case  367:  case  369:  case  371:  case  374:  case  375:  case  376:  case  377:  case  379:  case  380:  case  381:  case  382:
    return false;}
    tmp = getLeftModSubIn1Numbers(389,a,b,c,d,e);
    switch(tmp){  case 2:  case  3:  case  8:  case  10:  case  12:  case  14:  case  15:  case  18:  case  21:  case  22:  case  23:  case  26:  case  27:  case  29:  case  31:  case  32:  case  33:  case  34:  case  37:  case  38:  case  39:  case  40:  case  43:  case  47:  case  48:  case  50:  case  51:  case  53:  case  56:  case  57:  case  60:  case  61:  case  70:  case  71:  case  72:  case  75:  case  82:  case  83:  case  84:  case  88:  case  89:  case  90:  case  92:  case  98:  case  101:  case  103:  case  104:  case  105:  case  107:  case  108:  case  109:  case  110:  case  115:  case  116:  case  118:  case  123:  case  124:  case  126:  case  128:  case  130:  case  131:  case  132:  case  134:  case  135:  case  136:  case  138:  case  139:  case  145:  case  146:  case  147:  case  148:  case  149:  case  151:  case  152:  case  154:  case  155:  case  156:  case  158:  case  160:  case  161:  case  162:  case  163:  case  165:  case  167:  case  170:  case  172:  case  174:  case  177:  case  182:  case  185:  case  186:  case  188:  case  189:  case  190:  case  191:  case  192:  case  194:  case  195:  case  197:  case  198:  case  199:  case  200:  case  201:  case  203:  case  204:  case  207:  case  212:  case  215:  case  217:  case  219:  case  222:  case  224:  case  226:  case  227:  case  228:  case  229:  case  231:  case  233:  case  234:  case  235:  case  237:  case  238:  case  240:  case  241:  case  242:  case  243:  case  244:  case  250:  case  251:  case  253:  case  254:  case  255:  case  257:  case  258:  case  259:  case  261:  case  263:  case  265:  case  266:  case  271:  case  273:  case  274:  case  279:  case  280:  case  281:  case  282:  case  284:  case  285:  case  286:  case  288:  case  291:  case  297:  case  299:  case  300:  case  301:  case  305:  case  306:  case  307:  case  314:  case  317:  case  318:  case  319:  case  328:  case  329:  case  332:  case  333:  case  336:  case  338:  case  339:  case  341:  case  342:  case  346:  case  349:  case  350:  case  351:  case  352:
    return false;}
    tmp = getLeftModSubIn1Numbers(397,a,b,c,d,e);
    switch(tmp){  case 2:  case  3:  case  5:  case  6:  case  7:  case  8:  case  9:  case  10:  case  11:  case  12:  case  13:  case  14:  case  15:  case  17:  case  18:  case  19:  case  20:  case  21:  case  22:  case  23:  case  24:  case  25:  case  26:  case  28:  case  29:  case  32:  case  33:  case  36:  case  37:  case  38:  case  39:  case  40:  case  41:  case  42:  case  44:  case  45:  case  46:  case  47:  case  48:  case  49:  case  50:  case  51:  case  52:  case  53:  case  54:  case  55:  case  56:  case  57:  case  58:  case  59:  case  60:  case  61:  case  62:  case  63:  case  66:  case  67:  case  68:  case  69:  case  70:  case  71:  case  72:  case  73:  case  74:  case  75:  case  76:  case  77:  case  78:  case  79:  case  80:  case  81:  case  82:  case  84:  case  85:  case  86:  case  87:  case  88:  case  89:  case  90:  case  91:  case  92:  case  93:  case  94:  case  95:  case  96:  case  97:  case  98:  case  100:  case  101:  case  102:  case  103:  case  104:  case  105:  case  109:  case  111:  case  112:  case  113:  case  114:  case  115:  case  116:  case  117:  case  118:  case  119:  case  121:  case  122:  case  123:  case  125:  case  127:  case  128:  case  129:  case  130:  case  131:  case  132:  case  133:  case  134:  case  135:  case  138:  case  139:  case  143:  case  144:  case  145:  case  146:  case  148:  case  149:  case  150:  case  152:  case  153:  case  154:  case  155:  case  156:  case  157:  case  158:  case  159:  case  160:  case  161:  case  162:  case  164:  case  165:  case  166:  case  168:  case  169:  case  170:  case  173:  case  174:  case  175:  case  176:  case  177:  case  178:  case  179:  case  180:  case  181:  case  182:  case  183:  case  184:  case  185:  case  186:  case  187:  case  188:  case  189:  case  192:  case  193:  case  194:  case  195:  case  196:  case  197:  case  198:  case  199:  case  200:  case  201:  case  202:  case  203:  case  204:  case  205:  case  208:  case  209:  case  210:  case  211:  case  212:  case  213:  case  214:  case  215:  case  216:  case  217:  case  218:  case  219:  case  220:  case  221:  case  222:  case  223:  case  224:  case  227:  case  228:  case  229:  case  231:  case  232:  case  233:  case  235:  case  236:  case  237:  case  238:  case  239:  case  240:  case  241:  case  242:  case  243:  case  244:  case  245:  case  247:  case  248:  case  249:  case  251:  case  252:  case  253:  case  254:  case  258:  case  259:  case  262:  case  263:  case  264:  case  265:  case  266:  case  267:  case  268:  case  269:  case  270:  case  272:  case  274:  case  275:  case  276:  case  278:  case  279:  case  280:  case  281:  case  282:  case  283:  case  284:  case  285:  case  286:  case  288:  case  292:  case  293:  case  294:  case  295:  case  296:  case  297:  case  299:  case  300:  case  301:  case  302:  case  303:  case  304:  case  305:  case  306:  case  307:  case  308:  case  309:  case  310:  case  311:  case  312:  case  313:  case  315:  case  316:  case  317:  case  318:  case  319:  case  320:  case  321:  case  322:  case  323:  case  324:  case  325:  case  326:  case  327:  case  328:  case  329:  case  330:  case  331:  case  334:  case  335:  case  336:  case  337:  case  338:  case  339:  case  340:  case  341:  case  342:  case  343:  case  344:  case  345:  case  346:  case  347:  case  348:  case  349:  case  350:  case  351:  case  352:  case  353:  case  355:  case  356:  case  357:  case  358:  case  359:  case  360:  case  361:  case  364:  case  365:  case  368:  case  369:  case  371:  case  372:  case  373:  case  374:  case  375:  case  376:  case  377:  case  378:  case  379:  case  380:  case  382:  case  383:  case  384:  case  385:  case  386:  case  387:  case  388:  case  389:  case  390:  case  391:  case  392:  case  394:  case  395:
    return false;}
    tmp = getLeftModSubIn1Numbers(409,a,b,c,d,e);
    switch(tmp){  case 2:  case  3:  case  4:  case  7:  case  9:  case  10:  case  11:  case  12:  case  13:  case  14:  case  15:  case  16:  case  17:  case  18:  case  19:  case  20:  case  21:  case  22:  case  23:  case  24:  case  26:  case  28:  case  29:  case  31:  case  32:  case  33:  case  34:  case  35:  case  37:  case  38:  case  39:  case  41:  case  42:  case  43:  case  44:  case  45:  case  46:  case  47:  case  49:  case  50:  case  52:  case  53:  case  54:  case  55:  case  56:  case  57:  case  58:  case  59:  case  60:  case  61:  case  62:  case  63:  case  65:  case  66:  case  67:  case  70:  case  71:  case  72:  case  73:  case  74:  case  75:  case  76:  case  77:  case  78:  case  79:  case  80:  case  81:  case  84:  case  85:  case  86:  case  87:  case  88:  case  90:  case  91:  case  93:  case  94:  case  95:  case  96:  case  97:  case  98:  case  99:  case  100:  case  101:  case  102:  case  104:  case  105:  case  107:  case  108:  case  109:  case  110:  case  111:  case  112:  case  113:  case  114:  case  115:  case  116:  case  117:  case  118:  case  119:  case  120:  case  122:  case  123:  case  124:  case  126:  case  127:  case  128:  case  129:  case  130:  case  131:  case  132:  case  133:  case  134:  case  136:  case  137:  case  138:  case  139:  case  140:  case  141:  case  142:  case  144:  case  145:  case  146:  case  148:  case  149:  case  151:  case  152:  case  153:  case  155:  case  156:  case  157:  case  158:  case  159:  case  160:  case  161:  case  163:  case  164:  case  165:  case  166:  case  167:  case  168:  case  170:  case  171:  case  172:  case  173:  case  174:  case  175:  case  176:  case  177:  case  178:  case  179:  case  181:  case  183:  case  184:  case  185:  case  186:  case  187:  case  188:  case  189:  case  190:  case  191:  case  192:  case  194:  case  195:  case  197:  case  198:  case  199:  case  201:  case  202:  case  203:  case  204:  case  205:  case  206:  case  207:  case  208:  case  210:  case  211:  case  212:  case  214:  case  215:  case  217:  case  218:  case  219:  case  220:  case  221:  case  222:  case  223:  case  224:  case  225:  case  226:  case  228:  case  230:  case  231:  case  232:  case  233:  case  234:  case  235:  case  236:  case  237:  case  238:  case  239:  case  241:  case  242:  case  243:  case  244:  case  245:  case  246:  case  248:  case  249:  case  250:  case  251:  case  252:  case  253:  case  254:  case  256:  case  257:  case  258:  case  260:  case  261:  case  263:  case  264:  case  265:  case  267:  case  268:  case  269:  case  270:  case  271:  case  272:  case  273:  case  275:  case  276:  case  277:  case  278:  case  279:  case  280:  case  281:  case  282:  case  283:  case  285:  case  286:  case  287:  case  289:  case  290:  case  291:  case  292:  case  293:  case  294:  case  295:  case  296:  case  297:  case  298:  case  299:  case  300:  case  301:  case  302:  case  304:  case  305:  case  307:  case  308:  case  309:  case  310:  case  311:  case  312:  case  313:  case  314:  case  315:  case  316:  case  318:  case  319:  case  321:  case  322:  case  323:  case  324:  case  325:  case  328:  case  329:  case  330:  case  331:  case  332:  case  333:  case  334:  case  335:  case  336:  case  337:  case  338:  case  339:  case  342:  case  343:  case  344:  case  346:  case  347:  case  348:  case  349:  case  350:  case  351:  case  352:  case  353:  case  354:  case  355:  case  356:  case  357:  case  359:  case  360:  case  362:  case  363:  case  364:  case  365:  case  366:  case  367:  case  368:  case  370:  case  371:  case  372:  case  374:  case  375:  case  376:  case  377:  case  378:  case  380:  case  381:  case  383:  case  385:  case  386:  case  387:  case  388:  case  389:  case  390:  case  391:  case  392:  case  393:  case  394:  case  395:  case  396:  case  397:  case  398:  case  399:  case  400:  case  402:  case  405:  case  406:  case  407:
    return false;}
    tmp = getLeftModSubIn1Numbers(421,a,b,c,d,e);
    switch(tmp){  case 2:  case  3:  case  4:  case  5:  case  6:  case  8:  case  9:  case  10:  case  11:  case  12:  case  13:  case  14:  case  15:  case  16:  case  17:  case  18:  case  19:  case  20:  case  21:  case  22:  case  23:  case  24:  case  25:  case  26:  case  28:  case  29:  case  30:  case  31:  case  32:  case  34:  case  35:  case  37:  case  38:  case  39:  case  40:  case  41:  case  42:  case  43:  case  46:  case  47:  case  50:  case  52:  case  53:  case  54:  case  56:  case  57:  case  58:  case  59:  case  61:  case  62:  case  63:  case  65:  case  66:  case  69:  case  70:  case  71:  case  72:  case  73:  case  74:  case  76:  case  77:  case  79:  case  81:  case  82:  case  83:  case  84:  case  86:  case  87:  case  88:  case  89:  case  90:  case  91:  case  92:  case  93:  case  94:  case  95:  case  96:  case  97:  case  98:  case  99:  case  101:  case  102:  case  103:  case  105:  case  107:  case  108:  case  109:  case  110:  case  111:  case  112:  case  115:  case  116:  case  117:  case  118:  case  119:  case  120:  case  121:  case  122:  case  123:  case  124:  case  126:  case  127:  case  128:  case  129:  case  132:  case  133:  case  134:  case  135:  case  136:  case  137:  case  138:  case  140:  case  141:  case  143:  case  144:  case  145:  case  146:  case  147:  case  148:  case  149:  case  150:  case  151:  case  153:  case  154:  case  155:  case  156:  case  157:  case  158:  case  159:  case  160:  case  161:  case  162:  case  163:  case  164:  case  165:  case  166:  case  167:  case  168:  case  170:  case  171:  case  172:  case  173:  case  175:  case  176:  case  177:  case  178:  case  179:  case  180:  case  181:  case  182:  case  183:  case  184:  case  185:  case  186:  case  187:  case  188:  case  191:  case  192:  case  193:  case  194:  case  195:  case  196:  case  197:  case  198:  case  200:  case  201:  case  202:  case  203:  case  204:  case  205:  case  206:  case  207:  case  208:  case  209:  case  210:  case  211:  case  212:  case  213:  case  214:  case  215:  case  216:  case  217:  case  218:  case  219:  case  220:  case  221:  case  223:  case  224:  case  225:  case  226:  case  227:  case  228:  case  229:  case  230:  case  233:  case  234:  case  235:  case  236:  case  237:  case  238:  case  239:  case  240:  case  241:  case  242:  case  243:  case  244:  case  245:  case  246:  case  248:  case  249:  case  250:  case  251:  case  253:  case  254:  case  255:  case  256:  case  257:  case  258:  case  259:  case  260:  case  261:  case  262:  case  263:  case  264:  case  265:  case  266:  case  267:  case  268:  case  270:  case  271:  case  272:  case  273:  case  274:  case  275:  case  276:  case  277:  case  278:  case  280:  case  281:  case  283:  case  284:  case  285:  case  286:  case  287:  case  288:  case  289:  case  292:  case  293:  case  294:  case  295:  case  297:  case  298:  case  299:  case  300:  case  301:  case  302:  case  303:  case  304:  case  305:  case  306:  case  309:  case  310:  case  311:  case  312:  case  313:  case  314:  case  316:  case  318:  case  319:  case  320:  case  322:  case  323:  case  324:  case  325:  case  326:  case  327:  case  328:  case  329:  case  330:  case  331:  case  332:  case  333:  case  334:  case  335:  case  337:  case  338:  case  339:  case  340:  case  342:  case  344:  case  345:  case  347:  case  348:  case  349:  case  350:  case  351:  case  352:  case  355:  case  356:  case  358:  case  359:  case  360:  case  362:  case  363:  case  364:  case  365:  case  367:  case  368:  case  369:  case  371:  case  374:  case  375:  case  378:  case  379:  case  380:  case  381:  case  382:  case  383:  case  384:  case  386:  case  387:  case  389:  case  390:  case  391:  case  392:  case  393:  case  395:  case  396:  case  397:  case  398:  case  399:  case  400:  case  401:  case  402:  case  403:  case  404:  case  405:  case  406:  case  407:  case  408:  case  409:  case  410:  case  411:  case  412:  case  413:  case  415:  case  416:  case  417:  case  418:  case  419:
    return false;}
    tmp = getLeftModSubIn1Numbers(433,a,b,c,d,e);
    switch(tmp){  case 3:  case  5:  case  6:  case  7:  case  9:  case  10:  case  11:  case  12:  case  13:  case  14:  case  15:  case  17:  case  18:  case  19:  case  20:  case  21:  case  22:  case  23:  case  24:  case  25:  case  26:  case  28:  case  29:  case  30:  case  31:  case  33:  case  34:  case  36:  case  38:  case  39:  case  40:  case  41:  case  42:  case  43:  case  44:  case  45:  case  46:  case  47:  case  48:  case  49:  case  50:  case  51:  case  52:  case  53:  case  55:  case  56:  case  57:  case  58:  case  59:  case  60:  case  61:  case  62:  case  63:  case  65:  case  66:  case  67:  case  68:  case  69:  case  71:  case  72:  case  73:  case  76:  case  77:  case  78:  case  80:  case  81:  case  82:  case  83:  case  84:  case  85:  case  86:  case  87:  case  88:  case  89:  case  90:  case  91:  case  92:  case  93:  case  94:  case  95:  case  96:  case  97:  case  98:  case  100:  case  101:  case  102:  case  103:  case  104:  case  105:  case  106:  case  107:  case  109:  case  110:  case  111:  case  112:  case  113:  case  114:  case  116:  case  118:  case  119:  case  120:  case  121:  case  122:  case  123:  case  124:  case  125:  case  126:  case  129:  case  130:  case  131:  case  132:  case  134:  case  135:  case  136:  case  138:  case  139:  case  141:  case  142:  case  143:  case  144:  case  145:  case  146:  case  147:  case  149:  case  151:  case  152:  case  154:  case  155:  case  156:  case  157:  case  160:  case  161:  case  162:  case  163:  case  164:  case  165:  case  166:  case  168:  case  169:  case  170:  case  171:  case  172:  case  173:  case  174:  case  175:  case  176:  case  178:  case  180:  case  181:  case  182:  case  183:  case  184:  case  185:  case  186:  case  187:  case  188:  case  189:  case  190:  case  191:  case  192:  case  193:  case  194:  case  195:  case  196:  case  197:  case  200:  case  201:  case  202:  case  204:  case  205:  case  206:  case  207:  case  208:  case  209:  case  210:  case  211:  case  212:  case  213:  case  214:  case  215:  case  218:  case  219:  case  220:  case  221:  case  222:  case  223:  case  224:  case  225:  case  226:  case  227:  case  228:  case  229:  case  231:  case  232:  case  233:  case  236:  case  237:  case  238:  case  239:  case  240:  case  241:  case  242:  case  243:  case  244:  case  245:  case  246:  case  247:  case  248:  case  249:  case  250:  case  251:  case  252:  case  253:  case  255:  case  257:  case  258:  case  259:  case  260:  case  261:  case  262:  case  263:  case  264:  case  265:  case  267:  case  268:  case  269:  case  270:  case  271:  case  272:  case  273:  case  276:  case  277:  case  278:  case  279:  case  281:  case  282:  case  284:  case  286:  case  287:  case  288:  case  289:  case  290:  case  291:  case  292:  case  294:  case  295:  case  297:  case  298:  case  299:  case  301:  case  302:  case  303:  case  304:  case  307:  case  308:  case  309:  case  310:  case  311:  case  312:  case  313:  case  314:  case  315:  case  317:  case  319:  case  320:  case  321:  case  322:  case  323:  case  324:  case  326:  case  327:  case  328:  case  329:  case  330:  case  331:  case  332:  case  333:  case  335:  case  336:  case  337:  case  338:  case  339:  case  340:  case  341:  case  342:  case  343:  case  344:  case  345:  case  346:  case  347:  case  348:  case  349:  case  350:  case  351:  case  352:  case  353:  case  355:  case  356:  case  357:  case  360:  case  361:  case  362:  case  364:  case  365:  case  366:  case  367:  case  368:  case  370:  case  371:  case  372:  case  373:  case  374:  case  375:  case  376:  case  377:  case  378:  case  380:  case  381:  case  382:  case  383:  case  384:  case  385:  case  386:  case  387:  case  388:  case  389:  case  390:  case  391:  case  392:  case  393:  case  394:  case  395:  case  397:  case  399:  case  400:  case  402:  case  403:  case  404:  case  405:  case  407:  case  408:  case  409:  case  410:  case  411:  case  412:  case  413:  case  414:  case  415:  case  416:  case  418:  case  419:  case  420:  case  421:  case  422:  case  423:  case  424:  case  426:  case  427:  case  428:  case  430:
    return false;}
    tmp = getLeftModSubIn1Numbers(439,a,b,c,d,e);
    switch(tmp){  case 3:  case  5:  case  6:  case  10:  case  11:  case  12:  case  13:  case  15:  case  17:  case  19:  case  20:  case  21:  case  22:  case  23:  case  24:  case  25:  case  26:  case  27:  case  29:  case  30:  case  31:  case  33:  case  34:  case  35:  case  37:  case  38:  case  39:  case  40:  case  41:  case  42:  case  43:  case  44:  case  45:  case  46:  case  47:  case  48:  case  50:  case  51:  case  52:  case  53:  case  54:  case  57:  case  58:  case  59:  case  60:  case  62:  case  66:  case  67:  case  68:  case  69:  case  70:  case  71:  case  74:  case  75:  case  76:  case  77:  case  78:  case  79:  case  80:  case  82:  case  83:  case  84:  case  85:  case  86:  case  87:  case  88:  case  89:  case  90:  case  91:  case  92:  case  93:  case  94:  case  95:  case  96:  case  97:  case  99:  case  100:  case  101:  case  102:  case  104:  case  105:  case  106:  case  107:  case  108:  case  111:  case  113:  case  114:  case  115:  case  116:  case  117:  case  118:  case  119:  case  120:  case  121:  case  123:  case  124:  case  127:  case  129:  case  131:  case  132:  case  133:  case  134:  case  135:  case  136:  case  138:  case  139:  case  140:  case  142:  case  143:  case  147:  case  148:  case  149:  case  150:  case  151:  case  152:  case  153:  case  154:  case  155:  case  156:  case  157:  case  158:  case  159:  case  160:  case  161:  case  163:  case  164:  case  165:  case  166:  case  167:  case  168:  case  169:  case  170:  case  171:  case  172:  case  173:  case  174:  case  175:  case  176:  case  177:  case  178:  case  179:  case  180:  case  181:  case  182:  case  183:  case  184:  case  185:  case  186:  case  187:  case  188:  case  189:  case  190:  case  191:  case  192:  case  193:  case  194:  case  195:  case  197:  case  198:  case  199:  case  200:  case  201:  case  202:  case  203:  case  204:  case  205:  case  207:  case  208:  case  210:  case  211:  case  212:  case  213:  case  214:  case  215:  case  216:  case  217:  case  219:  case  221:
    return false;}
    tmp = getLeftModSubIn1Numbers(457,a,b,c,d,e);
    switch(tmp){  case 3:  case  5:  case  6:  case  7:  case  9:  case  10:  case  11:  case  12:  case  13:  case  14:  case  15:  case  18:  case  19:  case  20:  case  22:  case  23:  case  24:  case  26:  case  28:  case  29:  case  30:  case  31:  case  33:  case  35:  case  36:  case  37:  case  38:  case  39:  case  40:  case  41:  case  43:  case  44:  case  45:  case  46:  case  47:  case  48:  case  49:  case  51:  case  52:  case  53:  case  56:  case  58:  case  59:  case  60:  case  61:  case  62:  case  63:  case  65:  case  66:  case  67:  case  69:  case  70:  case  71:  case  72:  case  73:  case  74:  case  75:  case  76:  case  77:  case  78:  case  79:  case  80:  case  81:  case  82:  case  83:  case  85:  case  86:  case  88:  case  89:  case  90:  case  91:  case  92:  case  93:  case  94:  case  95:  case  96:  case  97:  case  98:  case  99:  case  101:  case  102:  case  103:  case  104:  case  105:  case  106:  case  107:  case  111:  case  112:  case  113:  case  115:  case  116:  case  117:  case  118:  case  119:  case  120:  case  122:  case  123:  case  124:  case  125:  case  126:  case  127:  case  129:  case  130:  case  131:  case  132:  case  133:  case  134:  case  135:  case  137:  case  138:  case  139:  case  140:  case  141:  case  142:  case  143:  case  144:  case  145:  case  146:  case  147:  case  148:  case  149:  case  150:  case  151:  case  152:  case  153:  case  154:  case  155:  case  156:  case  157:  case  158:  case  159:  case  160:  case  161:  case  162:  case  163:  case  164:  case  165:  case  166:  case  167:  case  169:  case  170:  case  171:  case  172:  case  173:  case  175:  case  176:  case  177:  case  178:  case  179:  case  180:  case  181:  case  182:  case  183:  case  184:  case  186:  case  187:  case  188:  case  189:  case  190:  case  191:  case  192:  case  193:  case  194:  case  195:  case  196:  case  197:  case  198:  case  199:  case  202:  case  203:  case  204:  case  205:  case  206:  case  207:  case  208:  case  209:  case  210:  case  211:  case  212:  case  213:  case  214:  case  217:  case  219:  case  221:  case  222:  case  223:  case  224:  case  225:  case  226:  case  227:  case  230:  case  231:  case  232:  case  233:  case  234:  case  235:  case  236:  case  238:  case  240:  case  243:  case  244:  case  245:  case  246:  case  247:  case  248:  case  249:  case  250:  case  251:  case  252:  case  253:  case  254:  case  255:  case  258:  case  259:  case  260:  case  261:  case  262:  case  263:  case  264:  case  265:  case  266:  case  267:  case  268:  case  269:  case  270:  case  271:  case  273:  case  274:  case  275:  case  276:  case  277:  case  278:  case  279:  case  280:  case  281:  case  282:  case  284:  case  285:  case  286:  case  287:  case  288:  case  290:  case  291:  case  292:  case  293:  case  294:  case  295:  case  296:  case  297:  case  298:  case  299:  case  300:  case  301:  case  302:  case  303:  case  304:  case  305:  case  306:  case  307:  case  308:  case  309:  case  310:  case  311:  case  312:  case  313:  case  314:  case  315:  case  316:  case  317:  case  318:  case  319:  case  320:  case  322:  case  323:  case  324:  case  325:  case  326:  case  327:  case  328:  case  330:  case  331:  case  332:  case  333:  case  334:  case  335:  case  337:  case  338:  case  339:  case  340:  case  341:  case  342:  case  344:  case  345:  case  346:  case  350:  case  351:  case  352:  case  353:  case  354:  case  355:  case  356:  case  358:  case  359:  case  360:  case  361:  case  362:  case  363:  case  364:  case  365:  case  366:  case  367:  case  368:  case  369:  case  371:  case  372:  case  374:  case  375:  case  376:  case  377:  case  378:  case  379:  case  380:  case  381:  case  382:  case  383:  case  384:  case  385:  case  386:  case  387:  case  388:  case  390:  case  391:  case  392:  case  394:  case  395:  case  396:  case  397:  case  398:  case  399:  case  401:  case  404:  case  405:  case  406:  case  408:  case  409:  case  410:  case  411:  case  412:  case  413:  case  414:  case  416:  case  417:  case  418:  case  419:  case  420:  case  421:  case  422:  case  424:  case  426:  case  427:  case  428:  case  429:  case  431:  case  433:  case  434:  case  435:  case  437:  case  438:  case  439:  case  442:  case  443:  case  444:  case  445:  case  446:  case  447:  case  448:  case  450:  case  451:  case  452:  case  454:
    return false;}
    tmp = getLeftModSubIn1Numbers(463,a,b,c,d,e);
    switch(tmp){  case 2:  case  3:  case  4:  case  5:  case  6:  case  7:  case  9:  case  10:  case  11:  case  12:  case  13:  case  14:  case  16:  case  17:  case  19:  case  20:  case  21:  case  22:  case  23:  case  24:  case  25:  case  26:  case  27:  case  28:  case  29:  case  30:  case  31:  case  32:  case  33:  case  35:  case  36:  case  37:  case  38:  case  39:  case  40:  case  41:  case  42:  case  43:  case  44:  case  45:  case  46:  case  48:  case  50:  case  51:  case  52:  case  53:  case  54:  case  56:  case  59:  case  60:  case  61:  case  62:  case  63:  case  67:  case  68:  case  69:  case  71:  case  72:  case  73:  case  74:  case  75:  case  76:  case  77:  case  79:  case  80:  case  81:  case  82:  case  83:  case  85:  case  87:  case  88:  case  89:  case  90:  case  91:  case  92:  case  93:  case  94:  case  95:  case  96:  case  98:  case  99:  case  101:  case  102:  case  103:  case  104:  case  105:  case  106:  case  107:  case  108:  case  109:  case  110:  case  112:  case  113:  case  114:  case  115:  case  116:  case  117:  case  119:  case  121:  case  122:  case  125:  case  126:  case  127:  case  128:  case  129:  case  130:  case  131:  case  132:  case  133:  case  135:  case  136:  case  137:  case  138:  case  139:  case  140:  case  141:  case  142:  case  143:  case  145:  case  147:  case  148:  case  150:  case  151:  case  152:  case  153:  case  154:  case  155:  case  156:  case  157:  case  160:  case  162:  case  163:  case  164:  case  165:  case  166:  case  167:  case  168:  case  169:  case  170:  case  171:  case  172:  case  173:  case  174:  case  175:  case  176:  case  177:  case  178:  case  179:  case  180:  case  181:  case  182:  case  183:  case  184:  case  185:  case  186:  case  187:  case  188:  case  190:  case  191:  case  192:  case  193:  case  194:  case  195:  case  196:  case  197:  case  198:  case  199:  case  200:  case  201:  case  202:  case  203:  case  204:  case  205:  case  206:  case  207:  case  208:  case  210:  case  211:  case  212:  case  213:  case  214:  case  215:  case  216:  case  217:  case  218:  case  219:  case  220:  case  221:  case  222:  case  223:  case  224:  case  227:  case  228:  case  229:  case  231:  case  232:  case  233:  case  234:  case  235:  case  236:  case  237:  case  238:  case  239:  case  240:  case  241:  case  243:  case  245:  case  246:  case  248:  case  249:  case  250:  case  251:  case  252:  case  253:  case  254:  case  255:  case  256:  case  257:  case  258:  case  259:  case  260:  case  261:  case  263:  case  264:  case  265:  case  267:  case  268:  case  269:  case  271:  case  273:  case  274:  case  275:  case  278:  case  280:  case  281:  case  282:  case  284:  case  285:  case  287:  case  288:  case  289:  case  290:  case  291:  case  292:  case  293:  case  294:  case  295:  case  296:  case  297:  case  298:  case  299:  case  300:  case  301:  case  302:  case  303:  case  304:  case  305:  case  306:  case  307:  case  309:  case  310:  case  311:  case  312:  case  314:  case  315:  case  316:  case  317:  case  318:  case  319:  case  320:  case  321:  case  322:  case  323:  case  325:  case  326:  case  327:  case  328:  case  329:  case  330:  case  331:  case  332:  case  333:  case  335:  case  336:  case  339:  case  340:  case  341:  case  342:  case  343:  case  344:  case  345:  case  347:  case  348:  case  349:  case  350:  case  351:  case  352:  case  353:  case  354:  case  355:  case  359:  case  360:  case  361:  case  363:  case  365:  case  366:  case  368:  case  369:  case  370:  case  371:  case  372:  case  373:  case  374:  case  375:  case  377:  case  378:  case  379:  case  382:  case  384:  case  385:  case  386:  case  387:  case  388:  case  390:  case  391:  case  393:  case  394:  case  395:  case  396:  case  397:  case  398:  case  399:  case  400:  case  401:  case  402:  case  403:  case  404:  case  405:  case  406:  case  408:  case  409:  case  410:  case  413:  case  414:  case  415:  case  416:  case  417:  case  418:  case  420:  case  421:  case  422:  case  423:  case  424:  case  426:  case  427:  case  428:  case  429:  case  430:  case  431:  case  432:  case  433:  case  434:  case  435:  case  437:  case  438:  case  439:  case  441:  case  442:  case  443:  case  444:  case  445:  case  446:  case  447:  case  448:  case  449:  case  450:  case  452:  case  454:  case  455:  case  457:  case  458:  case  459:  case  460:  case  461:  case  462:
    return false;}
    tmp = getLeftModSubIn1Numbers(487,a,b,c,d,e);
    switch(tmp){  case 2:  case  3:  case  4:  case  5:  case  6:  case  7:  case  9:  case  10:  case  11:  case  12:  case  13:  case  14:  case  15:  case  16:  case  17:  case  20:  case  21:  case  22:  case  23:  case  24:  case  26:  case  27:  case  28:  case  30:  case  31:  case  32:  case  33:  case  34:  case  36:  case  37:  case  38:  case  40:  case  42:  case  43:  case  44:  case  45:  case  46:  case  47:  case  48:  case  50:  case  52:  case  53:  case  54:  case  55:  case  56:  case  57:  case  58:  case  59:  case  62:  case  63:  case  65:  case  68:  case  69:  case  70:  case  71:  case  72:  case  73:  case  74:  case  75:  case  76:  case  77:  case  78:  case  79:  case  80:  case  81:  case  82:  case  83:  case  85:
    return false;}
    
    
    return true;
}



/*
__global__ 
void runCalcBCandDRange(int** BCAndDRange, int lowerB, int upperB, int aInt, int aPowerMod7){
    int index = blockIdx.x*blockDim.x + threadIdx.x;

    int bInt = upperB - index;


    int bPowerMod7 = (bInt % 7 == 0) ? 0 : 1;
    // This if is a filter precalculated. The sum of 4 6th power mod 7 cannot make 6
    int aMinusbMod7 = aPowerMod7 - bPowerMod7;
    while (aMinusbMod7 < 0){
        aMinusbMod7 += 7;
    }


    int mod7SumBCDEF = bPowerMod7;
    int lowerC = (int)(getLowerC(aInt, bInt) + 0.5);    //round up
    int upperC = (int)getUpperC(aInt, bInt);            //round down. But don't need it explicity
    upperC = (upperC > bInt) ? bInt : upperC;

    int* localResult;
    //malloc(&localResult, (upperC-lowerC+1)*sizeof(int)*4);
    localResult = (int*)malloc((upperC-lowerC+1)*sizeof(int)*4);
    //printf("%s\n", cudaGetErrorString(cudaGetLastError()));

    int cInt;
    int counter = 0;
    for(cInt = upperC; cInt > lowerC; cInt--){
        int cPowerMod7 = (cInt % 7 == 0) ? 0 : 1;
        mod7SumBCDEF = bPowerMod7 + cPowerMod7;
        if(mod7SumBCDEF > aPowerMod7){
            continue;
        }
        //printf("In c loop: aInt: %d, bInt: %d, cInt: %d, from block: %d, thread: %d\n", aInt, bInt, cInt, blockIdx.x, threadIdx.x);
        if (!isDecomposableIn3Numbers(aInt, bInt, cInt)){
            continue;
        }
        //printf("In c loop: aInt: %d, bInt: %d, cInt: %d, index: %d\n", aInt, bInt, cInt, index);
        int lowerD = (int)(getLowerD(aInt, bInt, cInt)+0.5);    //round up
        int upperD = (int) getUpperD(aInt, bInt, cInt);       //round down. But don't need it explicity

        upperD = (upperD > cInt) ? cInt : upperD;
        //bInt, cInt, lowerD, and UpperD

        
        //for(long long i = 0; i < 4LL*4*1024*1024*512; i += 4*4){
        //    BCAndDRange[i] = bInt;
        //    BCAndDRange[i+1] = cInt;
        //    BCAndDRange[i+2] = lowerD;
        //    BCAndDRange[i+3] = upperD;

        }
        
        
        localResult[counter*4] = bInt;
        localResult[counter*4+1] = cInt;
        localResult[counter*4+2] = lowerD;
        localResult[counter*4+3] = upperD;
        counter++;
    }

    //BCAndDRange[index] = localResult;
    //cudaMemcpy(BCAndDRange[index], localResult, index*4*4*counter, cudaMemcpyDeviceToDevice);
    //memcpy(BCAndDRange[index], localResult, index*4*4*counter, cudaMemcpyDeviceToDevice);
    memcpy(BCAndDRange[index], localResult, index*4*4*counter);
    
  
}
*/

__global__
void runMainComputing(int *d_BCAndDRange, long long counter, int aInt, int aPowerMod7){
    long long index = (blockIdx.x*blockDim.x + threadIdx.x)*4;
    //for(int i = 0; i < counter; i+=4){
    //d_BCAndDRange[i]      //bInt
    //d_BCAndDRange[i+1]    //cInt
    //d_BCAndDRange[i+2]    //lowerD
    //d_BCAndDRange[i+3]    //upperD
    
    int bInt = d_BCAndDRange[index];
    int cInt = d_BCAndDRange[index+1];
    int lowerD = d_BCAndDRange[index+2];
    int upperD = d_BCAndDRange[index+3];
    
    int bPowerMod7 = (bInt % 7 == 0) ? 0 : 1;
    int cPowerMod7 = (cInt % 7 == 0) ? 0 : 1;


    for(int dInt = upperD; dInt > lowerD; dInt--){
        int dPowerMod7 = (dInt % 7 == 0) ? 0 : 1;
        int mod7SumBCDEF = bPowerMod7 + cPowerMod7 + dPowerMod7;

        
        //////////////////////////////
        //cout << aInt << bInt << cInt << dInt << endl;

        if(mod7SumBCDEF >aPowerMod7){
            continue;
        }
        if(!isDecomposableIn2Numbers(aInt,bInt,cInt,dInt)){
            continue;
        }
        //printf("In d loop: aInt: %d, bInt: %d, cInt: %d, dInt: %d, from block: %d, thread: %d, index: %d\n", aInt, bInt, cInt, dInt, blockIdx.x, threadIdx.x, index);
        int lowerE = (int)(getLowerE(aInt, bInt, cInt, dInt)+0.5);    //round up
        int upperE = (int) getUpperE(aInt, bInt, cInt, dInt);       //round down. But don't need it explicity

        upperE = (upperE > dInt) ? dInt : upperE;

        int eInt;
        for(eInt = upperE; eInt > lowerE; eInt--){
            int ePowerMod7 = (eInt % 7 == 0) ? 0 : 1;
            mod7SumBCDEF = bPowerMod7 + cPowerMod7 + dPowerMod7 + ePowerMod7;
            if (mod7SumBCDEF > aPowerMod7){
                continue;
            }
            if (!isDecomposableIn1Numbers(aInt, bInt, cInt, dInt, eInt)){
                //cout << "isDecomposableIn1Numbers is not satisfied."<<endl;
                //cout << aInt<<", " << bInt<<", " << cInt<<", " << dInt<<", " << eInt<<", " << fInt<<", " << endl;
                continue;
            }
            //printf("In e loop: aInt: %d, bInt: %d, cInt: %d, dInt: %d, eInt: %d, from block: %d, thread: %d, index: %d\n", aInt, bInt, cInt, dInt, eInt, blockIdx.x, threadIdx.x, index);
            int upperF = (int)(getUpperF(aInt, bInt, cInt, dInt, eInt)+0.5);
            int lowerF = upperF - 1;

            upperF = (upperF > eInt) ? eInt : upperF;

            int fInt;
            for(fInt = upperF; fInt > lowerF; fInt--){
                
                if(!isABCDEFModEqual(aInt, bInt, cInt, dInt, eInt, fInt)){
                    //cout << "In isABCDEFModEqual:" << aInt<<", " << bInt<<", " << cInt<<", " << dInt<<", " << eInt<<", " << fInt<<", " << endl;
                    //printf("In isABCDEFModEqual (when not equal): aInt: %d, bInt: %d, cInt: %d, dInt: %d, eInt: %d, fInt: %d, from block: %d, thread: %d, index: %d, lowerF: %d, upperF: %d\n", aInt, bInt, cInt, dInt, eInt, fInt, blockIdx.x, threadIdx.x, index, lowerF, upperF);
                    continue;
                }
                
                printf("Found? aInt: %d, bInt: %d, cInt: %d, dInt: %d, eInt: %d, fInt: %d, from block: %d, thread: %d, index: %d\n", aInt, bInt, cInt, dInt, eInt, fInt, blockIdx.x, threadIdx.x, index);
            }
        }
    }
    //printf("Done with thread. In d loop: aInt: %d, bInt: %d, cInt: %d, from block: %d, thread: %d, index: %lld\n", aInt, bInt, cInt, blockIdx.x, threadIdx.x, index);
    //}
}

int main(void){
    //int aInt = 1234567;
    int aInt = 12345;
    //int aInt = 123456;
    //int primes1000[] = {2, 3, 5, 7, 11, 13, 17, 19, 23, 29, 31, 37, 41, 43, 47, 53, 59, 61, 67, 71, 73, 79, 83, 89, 97, 101, 103, 107, 109, 113, 127, 131, 137, 139, 149, 151, 157, 163, 167, 173, 179, 181, 191, 193, 197, 199, 211, 223, 227, 229, 233, 239, 241, 251, 257, 263, 269, 271, 277, 281, 283, 293, 307, 311, 313, 317, 331, 337, 347, 349, 353, 359, 367, 373, 379, 383, 389, 397, 401, 409, 419, 421, 431, 433, 439, 443, 449, 457, 461, 463, 467, 479, 487, 491, 499, 503, 509, 521, 523, 541, 547, 557, 563, 569, 571, 577, 587, 593, 599, 601, 607, 613, 617, 619, 631, 641, 643, 647, 653, 659, 661, 673, 677, 683, 691, 701, 709, 719, 727, 733, 739, 743, 751, 757, 761, 769, 773, 787, 797, 809, 811, 821, 823, 827, 829, 839, 853, 857, 859, 863, 877, 881, 883, 887, 907, 911, 919, 929, 937, 941, 947, 953, 967, 971, 977, 983, 991, 997};

    //int *d_primes1000;
    //cudaMallocManaged(&d_primes1000, sizeof(primes1000));
    //cudaDeviceSynchronize();
    //printf("%s\n", cudaGetErrorString(cudaGetLastError()));

    //cudaMemcpy(d_primes1000, &primes1000, sizeof(primes1000), cudaMemcpyHostToDevice);
    //cudaDeviceSynchronize();
    //printf("%s\n", cudaGetErrorString(cudaGetLastError()));
    int *BCAndDRange;
    long long sizeLimit = 10*(1LL<<30);    // (1LL<<30) Bytes = 1GB
    BCAndDRange = (int*)malloc(sizeLimit);

    int *d_BCAndDRange;
    hipMalloc(&d_BCAndDRange, sizeLimit);        
    printf("%s\n", hipGetErrorString(hipGetLastError()));


    for(int n = aInt; n < aInt + 1; n++){
        int lowerB = (int)(aInt/pow(5, 1.0f/6) + 0.5); 
        int upperB = aInt - 1;

        int aPowerMod7 = (aInt % 7 == 0) ? 0 : 1;
        if(aPowerMod7 == 0)continue;

        //int numberOfthreads = 256;
        //int numberOfthreads = 4;
        printf("lowerB: %d, upperB: %d\n", lowerB, upperB);
       
        //int **BCAndDRange;
        //cudaMalloc(&BCAndDRange, (upperB-lowerB+1)*sizeof(int*));
        


        //int *d_BCAndDRange;
        //cudaMalloc(&d_BCAndDRange, sizeof(int)*4LL*(1024*1024*512));     //total amount is 8GB for sizeof(int)*4*(1024*1024*512)
        //printf("%s\n", cudaGetErrorString(cudaGetLastError()));

        ////////////////////////////////////////////////
        
        long long counter = 0;

        for(int bInt = upperB; bInt >= lowerB; bInt--){
            if(counter > (sizeLimit/4)){
                    cout<<"Inner Loop. ";
                    cout<<"counter: "<<counter;
                    cout<<". counterLimit: "<<(sizeLimit/4)<<endl;
                    break;
                }
            int bPowerMod7 = (bInt % 7 == 0) ? 0 : 1;
            // This if is a filter precalculated. The sum of 4 6th power mod 7 cannot make 6
            int aMinusbMod7 = aPowerMod7 - bPowerMod7;
            while (aMinusbMod7 < 0){
                aMinusbMod7 += 7;
            }
        
        
            int mod7SumBCDEF = bPowerMod7;
            int lowerC = (int)(getLowerC(aInt, bInt) + 0.5);    //round up
            int upperC = (int)getUpperC(aInt, bInt);            //round down. But don't need it explicity
            upperC = (upperC > bInt) ? bInt : upperC;

            int cInt;
            for(cInt = upperC; cInt > lowerC; cInt--){
                int cPowerMod7 = (cInt % 7 == 0) ? 0 : 1;
                mod7SumBCDEF = bPowerMod7 + cPowerMod7;
                if(mod7SumBCDEF > aPowerMod7){
                    continue;
                }
                //printf("In c loop: aInt: %d, bInt: %d, cInt: %d, from block: %d, thread: %d\n", aInt, bInt, cInt, blockIdx.x, threadIdx.x);
                if (!isDecomposableIn3Numbers(aInt, bInt, cInt)){
                    continue;
                }


                //printf("In c loop: aInt: %d, bInt: %d, cInt: %d, index: %d\n", aInt, bInt, cInt, index);
                int lowerD = (int)(getLowerD(aInt, bInt, cInt)+0.5);    //round up
                int upperD = (int) getUpperD(aInt, bInt, cInt);       //round down. But don't need it explicity

                upperD = (upperD > cInt) ? cInt : upperD;

                BCAndDRange[counter] = bInt;
                BCAndDRange[counter+1] = cInt;
                BCAndDRange[counter+2] = lowerD;
                BCAndDRange[counter+3] = upperD;
                counter+=4;
                if(counter > (sizeLimit/4)){
                    cout<<"Inner Loop. ";
                    cout<<"counter: "<<counter;
                    cout<<". counterLimit: "<<(sizeLimit/4)<<endl;
                    cout<<"bInt: "<<bInt<<", ";
                    cout<<"cInt: "<<cInt<<", ";
                    cout<<"lowerD: "<<lowerD<<", ";
                    cout<<"upperD: "<<lowerD<<endl;

                    cout<<"counter output 1: "<<counter<<endl;
                    

                    hipMemcpy(d_BCAndDRange, BCAndDRange, sizeLimit, hipMemcpyHostToDevice);
                    printf("%s\n", hipGetErrorString(hipGetLastError()));

                    //////////////////////////////////////////
                    ///// call kernel
                    //long long amount = counter/4;
                    runMainComputing<<<counter/4, threadAmount>>>(d_BCAndDRange, counter, aInt, aPowerMod7);
                    hipDeviceSynchronize();
                    //////////////////////////////////////////
                    
                    counter=0;
                    continue;
                }
            }
        }


        //this if means the last portion is less than whole memory block
        if (counter != 0 && counter < sizeLimit/4){
            hipMemcpy(d_BCAndDRange, BCAndDRange, counter * 4, hipMemcpyHostToDevice);
            printf("%s\n", hipGetErrorString(hipGetLastError()));
            //////////////////////////////////////////
            ///// call kernel
            //long long amount = counter/4;
            printf("Starting runMainComputing Threads.\n");
            runMainComputing<<<counter/4, threadAmount>>>(d_BCAndDRange, counter, aInt, aPowerMod7);
            //////////////////////////////////////////
        }
        ////////////////////////////////////////////////




        
        //cout<<"Pause Using cin"<<endl;
        //cout<<"counter: "<<counter<<endl;
        //int age;
        //cin >> age;

        hipFree(d_BCAndDRange);
        printf("%s\n", hipGetErrorString(hipGetLastError()));

        //cudaDeviceSynchronize();
        free(BCAndDRange);
        printf("%s\n", hipGetErrorString(hipGetLastError()));

        return 0;
    }
}
